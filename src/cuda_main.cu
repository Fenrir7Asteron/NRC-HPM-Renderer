#include <engine/cuda_common.hpp>
#include <engine/util/Log.hpp>
#include <engine/graphics/Window.hpp>
#include <engine/graphics/VulkanAPI.hpp>
#include <engine/graphics/vulkan/CommandPool.hpp>
#include <engine/graphics/vulkan/CommandRecorder.hpp>
#include <engine/graphics/renderer/ImGuiRenderer.hpp>
#include <engine/graphics/vulkan/Swapchain.hpp>
#include <imgui.h>
#include <engine/graphics/renderer/NrcHpmRenderer.hpp>
#include <engine/graphics/NeuralRadianceCache.hpp>
#include <engine/util/read_file.hpp>
#include <engine/util/Input.hpp>
#include <engine/util/Time.hpp>
#include <engine/HpmScene.hpp>
#include <engine/AppConfig.hpp>
#include <filesystem>
#include <engine/graphics/renderer/McHpmRenderer.hpp>

en::NrcHpmRenderer* nrcHpmRenderer = nullptr;
en::McHpmRenderer* mcHpmRenderer = nullptr;

void RecordSwapchainCommandBuffer(VkCommandBuffer commandBuffer, VkImage image)
{
	uint32_t width = en::Window::GetWidth();
	uint32_t height = en::Window::GetHeight();

	VkCommandBufferBeginInfo beginInfo;
	beginInfo.sType = VK_STRUCTURE_TYPE_COMMAND_BUFFER_BEGIN_INFO;
	beginInfo.pNext = nullptr;
	beginInfo.flags = 0;
	beginInfo.pInheritanceInfo = nullptr;

	VkResult result = vkBeginCommandBuffer(commandBuffer, &beginInfo);
	if (result != VK_SUCCESS)
		en::Log::Error("Failed to begin VkCommandBuffer", true);

	en::vk::CommandRecorder::ImageLayoutTransfer(
		commandBuffer,
		image,
		VK_IMAGE_LAYOUT_UNDEFINED,
		VK_IMAGE_LAYOUT_TRANSFER_DST_OPTIMAL,
		VK_ACCESS_NONE_KHR,
		VK_ACCESS_TRANSFER_WRITE_BIT,
		VK_PIPELINE_STAGE_TOP_OF_PIPE_BIT,
		VK_PIPELINE_STAGE_TRANSFER_BIT);

	if (nrcHpmRenderer != nullptr && mcHpmRenderer != nullptr && en::ImGuiRenderer::IsInitialized())
	{
		VkImageCopy imageCopy;
		imageCopy.srcSubresource.aspectMask = VK_IMAGE_ASPECT_COLOR_BIT;
		imageCopy.srcSubresource.mipLevel = 0;
		imageCopy.srcSubresource.baseArrayLayer = 0;
		imageCopy.srcSubresource.layerCount = 1;
		imageCopy.srcOffset = { 0, 0, 0 };
		imageCopy.dstSubresource.aspectMask = VK_IMAGE_ASPECT_COLOR_BIT;
		imageCopy.dstSubresource.mipLevel = 0;
		imageCopy.dstSubresource.baseArrayLayer = 0;
		imageCopy.dstSubresource.layerCount = 1;
		imageCopy.dstOffset = { 0, 0, 0 };
		imageCopy.extent = { width, height, 1 };

		vkCmdCopyImage(
			commandBuffer,
			en::ImGuiRenderer::GetImage(),
			VK_IMAGE_LAYOUT_TRANSFER_SRC_OPTIMAL,
			image,
			VK_IMAGE_LAYOUT_TRANSFER_DST_OPTIMAL,
			1,
			&imageCopy);
	}

	en::vk::CommandRecorder::ImageLayoutTransfer(
		commandBuffer,
		image,
		VK_IMAGE_LAYOUT_TRANSFER_DST_OPTIMAL,
		VK_IMAGE_LAYOUT_PRESENT_SRC_KHR,
		VK_ACCESS_TRANSFER_WRITE_BIT,
		VK_ACCESS_COLOR_ATTACHMENT_READ_BIT,
		VK_PIPELINE_STAGE_TRANSFER_BIT,
		VK_PIPELINE_STAGE_COLOR_ATTACHMENT_OUTPUT_BIT);

	result = vkEndCommandBuffer(commandBuffer);
	if (result != VK_SUCCESS)
		en::Log::Error("Failed to end VkCommandBuffer", true);
}

void SwapchainResizeCallback()
{
	en::Window::WaitForUsableSize();
	vkDeviceWaitIdle(en::VulkanAPI::GetDevice()); // TODO: causes error with multithreaded rendering

	en::Log::Info("Skipping swapchain resize callback");

	//uint32_t width = en::Window::GetWidth();
	//uint32_t height = en::Window::GetHeight();
	//nrcHpmRenderer->ResizeFrame(width, height);
	//en::ImGuiRenderer::Resize(width, height);
	//en::ImGuiRenderer::SetBackgroundImageView(imageView);
}

void Benchmark(
	uint32_t width, 
	uint32_t height, 
	uint32_t sceneID, 
	const en::AppConfig& appConfig, 
	const en::HpmScene& scene, 
	VkQueue queue)
{
	// Create benchmark camera
	const float aspectRatio = static_cast<float>(width) / static_cast<float>(height);

	std::array<en::Camera, 6> cameras = {
		en::Camera(
			glm::vec3(64.0f, 0.0f, 0.0f),
			glm::vec3(-1.0f, 0.0f, 0.0f),
			glm::vec3(0.0f, 1.0f, 0.0f),
			aspectRatio,
			glm::radians(60.0f),
			0.1f,
			100.0f),
		en::Camera(
			glm::vec3(-64.0f, 0.0f, 0.0f),
			glm::vec3(1.0f, 0.0f, 0.0f),
			glm::vec3(0.0f, 1.0f, 0.0f),
			aspectRatio,
			glm::radians(60.0f),
			0.1f,
			100.0f),
		en::Camera(
			glm::vec3(0.0f, 64.0f, 0.0f),
			glm::vec3(0.0f, -1.0f, 0.0f),
			glm::vec3(1.0f, 0.0f, 0.0f),
			aspectRatio,
			glm::radians(60.0f),
			0.1f,
			100.0f),
		en::Camera(
			glm::vec3(0.0f, -64.0f, 0.0f),
			glm::vec3(0.0f, 1.0f, 0.0f),
			glm::vec3(1.0f, 0.0f, 0.0f),
			aspectRatio,
			glm::radians(60.0f),
			0.1f,
			100.0f),
		en::Camera(
			glm::vec3(0.0f, 0.0f, 64.0f),
			glm::vec3(0.0f, 0.0f, -1.0f),
			glm::vec3(0.0f, 1.0f, 0.0f),
			aspectRatio,
			glm::radians(60.0f),
			0.1f,
			100.0f),
		en::Camera(
			glm::vec3(0.0f, 0.0f, -64.0f),
			glm::vec3(0.0f, 0.0f, 1.0f),
			glm::vec3(0.0f, 1.0f, 0.0f),
			aspectRatio,
			glm::radians(60.0f),
			0.1f,
			100.0f),
	};

	// Create output path if not exists
	std::string outputDirPath = "output/ " + appConfig.GetName() + "/";
	if (!std::filesystem::is_directory(outputDirPath) || !std::filesystem::exists(outputDirPath))
	{
		std::filesystem::create_directory(outputDirPath);
	}

	// Create reference folder if not exists
	std::string referenceDirPath = "output/" + std::to_string(sceneID) + "/";
#if __cplusplic >= 201703L
	en::Log::Warn("C++ version lower then 17. Cant create reference data");
#else
	if (!std::filesystem::is_directory(referenceDirPath) || !std::filesystem::exists(referenceDirPath))
	{
		en::Log::Info("Reference folder for scene " + std::to_string(sceneID) + " was not found. Creating reference images");

		// Create folder
		std::filesystem::create_directory(referenceDirPath);

		// Create ground truth renderer
		en::McHpmRenderer* gtRenderer = nullptr;

		for (size_t i = 0; i < cameras.size(); i++)
		{
			en::Log::Info("Generating reference image " + std::to_string(i));

			// Set new camera
			if (gtRenderer == nullptr) { gtRenderer = new en::McHpmRenderer(width, height, 64, &cameras[i], scene); }
			else { gtRenderer->SetCamera(&cameras[i]); }

			// Generate reference image
			for (size_t frame = 0; frame < 1024; frame++)
			{
				gtRenderer->Render(queue);
				ASSERT_VULKAN(vkQueueWaitIdle(queue));
			}

			// Export reference image
			gtRenderer->ExportOutputImageToFile(queue, referenceDirPath + std::to_string(i) + ".exr");
		}

		// Destroy resources
		gtRenderer->Destroy();
		delete gtRenderer;
	}
#endif

	// Test frame
	for (size_t i = 0; i < cameras.size(); i++)
	{
		nrcHpmRenderer->SetCamera(&cameras[i]);
		nrcHpmRenderer->Render(queue);
		ASSERT_VULKAN(vkQueueWaitIdle(queue));
		nrcHpmRenderer->ExportOutputImageToFile(queue, outputDirPath + "nrc_" + std::to_string(i) + ".exr");
	
		mcHpmRenderer->SetCamera(&cameras[i]);
		mcHpmRenderer->Render(queue);
		ASSERT_VULKAN(vkQueueWaitIdle(queue));
		mcHpmRenderer->ExportOutputImageToFile(queue, outputDirPath + "mc_" + std::to_string(i) + ".exr");
	}

	// Destroy resources
	for (size_t i = 0; i < cameras.size(); i++) { cameras[i].Destroy(); }
}

bool RunAppConfigInstance(const en::AppConfig& appConfig)
{
	// Start engine
	const std::string appName("NRC-HPM-Renderer");
	uint32_t width = appConfig.renderWidth;
	uint32_t height = appConfig.renderHeight;
	en::Log::Info("Starting " + appName);
	en::Window::Init(width, height, false, appName);
	en::Input::Init(en::Window::GetGLFWHandle());
	en::VulkanAPI::Init(appName);
	const VkDevice device = en::VulkanAPI::GetDevice();
	const uint32_t qfi = en::VulkanAPI::GetGraphicsQFI();
	const VkQueue queue = en::VulkanAPI::GetGraphicsQueue();

	// Renderer select
	const std::vector<char*> rendererMenuItems = { "MC", "NRC" }; // TODO: Restir
	const char* currentRendererMenuItem = rendererMenuItems[1];
	uint32_t rendererId = 1;

	// Init resources
	en::NeuralRadianceCache nrc(appConfig);

	en::HpmScene hpmScene(appConfig);

	// Setup rendering
	en::Camera camera(
		glm::vec3(64.0f, 0.0f, 0.0f),
		glm::vec3(-1.0f, 0.0f, 0.0f),
		glm::vec3(0.0f, 1.0f, 0.0f),
		static_cast<float>(width) / static_cast<float>(height),
		glm::radians(60.0f),
		0.1f,
		100.0f);

	// Init rendering pipeline
	en::vk::Swapchain swapchain(width, height, RecordSwapchainCommandBuffer, SwapchainResizeCallback);

	nrcHpmRenderer = new en::NrcHpmRenderer(
		width,
		height,
		appConfig.trainSampleRatio,
		appConfig.trainSpp,
		&camera,
		hpmScene,
		nrc);

	mcHpmRenderer = new en::McHpmRenderer(width, height, 32, &camera, hpmScene);

	en::ImGuiRenderer::Init(width, height);
	switch (rendererId)
	{
	case 0: // MC
		en::ImGuiRenderer::SetBackgroundImageView(mcHpmRenderer->GetImageView());
		break;
	case 1: // NRC
		en::ImGuiRenderer::SetBackgroundImageView(nrcHpmRenderer->GetImageView());
		break;
	default: // Error
		en::Log::Error("Renderer ID is invalid", true);
		break;
	}

	// Swapchain rerecording because imgui renderer is now available
	swapchain.Resize(width, height);

	// Main loop
	VkResult result;
	size_t frameCount = 0;
	bool shutdown = false;
	bool restartAfterClose = false;
	while (!en::Window::IsClosed() && !shutdown)
	{
		// Exit
		//if (frameCount == 10) { break; }

		// Update
		en::Window::Update();
		en::Input::Update();
		en::Time::Update();

		width = en::Window::GetWidth();
		height = en::Window::GetHeight();

		float deltaTime = static_cast<float>(en::Time::GetDeltaTime());
		uint32_t fps = en::Time::GetFps();

		// Physics
		en::Input::HandleUserCamInput(&camera, deltaTime);
		camera.SetAspectRatio(width, height);
		camera.UpdateUniformBuffer();

		// Render
		// Always render nrc for training
		//nrcHpmRenderer->Render(queue);
		//result = vkQueueWaitIdle(queue);
		//ASSERT_VULKAN(result);
		//nrcHpmRenderer->EvaluateTimestampQueries();

		switch (rendererId)
		{
		case 0: // MC
			mcHpmRenderer->Render(queue);
			result = vkQueueWaitIdle(queue);
			ASSERT_VULKAN(result);
			mcHpmRenderer->EvaluateTimestampQueries();
			break;
		case 1: // NRC
			nrcHpmRenderer->Render(queue);
			result = vkQueueWaitIdle(queue);
			ASSERT_VULKAN(result);
			nrcHpmRenderer->EvaluateTimestampQueries();
			break;
		default: // Error
			en::Log::Error("Renderer ID is invalid", true);
			break;
		}

		// Imgui
		en::ImGuiRenderer::StartFrame();

		ImGui::Begin("Statistics");
		ImGui::Text((std::string("Framecount ") + std::to_string(frameCount)).c_str());
		ImGui::Text("DeltaTime %f", deltaTime);
		ImGui::Text("FPS %d", fps);
		ImGui::Text("NRC Loss %f", nrc.GetLoss());
		ImGui::End();

		ImGui::Begin("Controls");
		shutdown = ImGui::Button("Shutdown");
		ImGui::Checkbox("Restart after shutdown", &restartAfterClose);
		
		if (ImGui::BeginCombo("##combo", currentRendererMenuItem))
		{
			for (int i = 0; i < rendererMenuItems.size(); i++)
			{
				bool selected = (currentRendererMenuItem == rendererMenuItems[i]);
				if (ImGui::Selectable(rendererMenuItems[i], selected))
				{
					if (i != rendererId)
					{
						rendererId = i;
						switch (rendererId)
						{
						case 0: // MC
							en::ImGuiRenderer::SetBackgroundImageView(mcHpmRenderer->GetImageView());
							break;
						case 1: // NRC
							en::ImGuiRenderer::SetBackgroundImageView(nrcHpmRenderer->GetImageView());
							break;
						default: // Error
							en::Log::Error("Renderer ID is invalid", true);
							break;
						}
					}
					currentRendererMenuItem = rendererMenuItems[i]; 
				};
				if (selected) { ImGui::SetItemDefaultFocus(); }
			}
			ImGui::EndCombo();
		}

		ImGui::End();

		mcHpmRenderer->RenderImGui();
		nrcHpmRenderer->RenderImGui();

		hpmScene.Update(true);

		appConfig.RenderImGui();

		en::ImGuiRenderer::EndFrame(queue, VK_NULL_HANDLE);
		result = vkQueueWaitIdle(queue);
		ASSERT_VULKAN(result);

		// Display
		swapchain.DrawAndPresent(VK_NULL_HANDLE, VK_NULL_HANDLE);
		frameCount++;
	}

	// Evaluate at end
	Benchmark(appConfig.renderWidth, appConfig.renderHeight, appConfig.scene.id, appConfig, hpmScene, queue);

	// Stop gpu work
	result = vkDeviceWaitIdle(device);
	ASSERT_VULKAN(result);

	// End
	mcHpmRenderer->Destroy();
	delete mcHpmRenderer;
	
	nrcHpmRenderer->Destroy();
	delete nrcHpmRenderer;
	en::ImGuiRenderer::Shutdown();
	swapchain.Destroy(true);

	hpmScene.Destroy();
	camera.Destroy();
	nrc.Destroy();

	en::VulkanAPI::Shutdown();
	en::Window::Shutdown();
	en::Log::Info("Ending " + appName);

	return restartAfterClose;
}

int main(int argc, char** argv)
{
	en::AppConfig appConfig(argc, argv);

	bool restartRunConfig;
	do {
		restartRunConfig = RunAppConfigInstance(appConfig);
	} while (restartRunConfig);

	return 0;
}
