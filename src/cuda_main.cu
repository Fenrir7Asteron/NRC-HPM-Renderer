#include <cuda_main.hpp>

#define VK_USE_PLATFORM_WIN32_KHR

#include <aclapi.h>
#include <dxgi1_2.h>
#include <windows.h>
#include <VersionHelpers.h>

#include <engine/util/Log.hpp>
#include <engine/graphics/Window.hpp>
#include <engine/graphics/VulkanAPI.hpp>
#include <engine/graphics/vulkan/CommandPool.hpp>
#include <engine/graphics/vulkan/CommandRecorder.hpp>

#include <tiny-cuda-nn/config.h>
#include <vulkan/vulkan.h>

#define ASSERT_CUDA(error) if (error != hipSuccess) { en::Log::Error("Cuda assert triggered: " + std::string(hipGetErrorName(error)), true); }

PFN_vkGetMemoryWin32HandleKHR fpGetMemoryWin32HandleKHR;
PFN_vkGetSemaphoreWin32HandleKHR fpGetSemaphoreWin32HandleKHR;

VkImage image;
VkDeviceMemory imageMemory;

VkExternalMemoryHandleTypeFlagBits externalMemoryHandleType = VK_EXTERNAL_MEMORY_HANDLE_TYPE_OPAQUE_WIN32_BIT;

void LoadVulkanProcAddr()
{
	fpGetMemoryWin32HandleKHR = (PFN_vkGetMemoryWin32HandleKHR)vkGetInstanceProcAddr(
		en::VulkanAPI::GetInstance(), 
		"vkGetMemoryWin32HandleKHR");

	fpGetSemaphoreWin32HandleKHR = (PFN_vkGetSemaphoreWin32HandleKHR)vkGetDeviceProcAddr(
		en::VulkanAPI::GetDevice(),
		"vkGetSemaphoreWin32HandleKHR");
}

void CreateImage(uint32_t width, uint32_t height)
{
	VkFormat format = VK_FORMAT_R32G32B32A32_SFLOAT;
	VkDevice device = en::VulkanAPI::GetDevice();

	// Create Image
	VkExternalMemoryImageCreateInfo vkExternalMemImageCreateInfo = {};
	vkExternalMemImageCreateInfo.sType = VK_STRUCTURE_TYPE_EXTERNAL_MEMORY_IMAGE_CREATE_INFO;
	vkExternalMemImageCreateInfo.pNext = nullptr;
	vkExternalMemImageCreateInfo.handleTypes = externalMemoryHandleType;

	VkImageCreateInfo imageCI;
	imageCI.sType = VK_STRUCTURE_TYPE_IMAGE_CREATE_INFO;
	imageCI.pNext = &vkExternalMemImageCreateInfo;
	imageCI.flags = 0;
	imageCI.imageType = VK_IMAGE_TYPE_2D;
	imageCI.format = format;
	imageCI.extent = { width, height, 1 };
	imageCI.mipLevels = 1;
	imageCI.arrayLayers = 1;
	imageCI.samples = VK_SAMPLE_COUNT_1_BIT;
	imageCI.tiling = VK_IMAGE_TILING_OPTIMAL;
	imageCI.usage = VK_IMAGE_USAGE_STORAGE_BIT | VK_IMAGE_USAGE_TRANSFER_SRC_BIT | VK_IMAGE_USAGE_TRANSFER_DST_BIT;
	imageCI.sharingMode = VK_SHARING_MODE_EXCLUSIVE;
	imageCI.queueFamilyIndexCount = 0;
	imageCI.pQueueFamilyIndices = nullptr;
	imageCI.initialLayout = VK_IMAGE_LAYOUT_UNDEFINED;

	VkResult result = vkCreateImage(device, &imageCI, nullptr, &image);
	ASSERT_VULKAN(result);

	// Image Memory
	SECURITY_ATTRIBUTES winSecurityAttributes{};
	
	VkExportMemoryWin32HandleInfoKHR vulkanExportMemoryWin32HandleInfoKHR = {};
	vulkanExportMemoryWin32HandleInfoKHR.sType = VK_STRUCTURE_TYPE_EXPORT_MEMORY_WIN32_HANDLE_INFO_KHR;
	vulkanExportMemoryWin32HandleInfoKHR.pNext = NULL;
	vulkanExportMemoryWin32HandleInfoKHR.pAttributes = &winSecurityAttributes;
	vulkanExportMemoryWin32HandleInfoKHR.dwAccess = DXGI_SHARED_RESOURCE_READ | DXGI_SHARED_RESOURCE_WRITE;
	vulkanExportMemoryWin32HandleInfoKHR.name = (LPCWSTR)NULL;

	VkExportMemoryAllocateInfoKHR vulkanExportMemoryAllocateInfoKHR = {};
	vulkanExportMemoryAllocateInfoKHR.sType = VK_STRUCTURE_TYPE_EXPORT_MEMORY_ALLOCATE_INFO_KHR;
	vulkanExportMemoryAllocateInfoKHR.pNext = &vulkanExportMemoryWin32HandleInfoKHR;
	vulkanExportMemoryAllocateInfoKHR.handleTypes = externalMemoryHandleType;
	
	VkMemoryRequirements memoryRequirements;
	vkGetImageMemoryRequirements(device, image, &memoryRequirements);

	VkMemoryAllocateInfo allocateInfo;
	allocateInfo.sType = VK_STRUCTURE_TYPE_MEMORY_ALLOCATE_INFO;
	allocateInfo.pNext = &vulkanExportMemoryAllocateInfoKHR;
	allocateInfo.allocationSize = memoryRequirements.size;
	allocateInfo.memoryTypeIndex = en::VulkanAPI::FindMemoryType(
		memoryRequirements.memoryTypeBits,
		VK_MEMORY_PROPERTY_DEVICE_LOCAL_BIT);

	result = vkAllocateMemory(device, &allocateInfo, nullptr, &imageMemory);
	ASSERT_VULKAN(result);

	result = vkBindImageMemory(device, image, imageMemory, 0);
	ASSERT_VULKAN(result);

	// Change image layout
	en::vk::CommandPool commandPool(0, en::VulkanAPI::GetGraphicsQFI());
	commandPool.AllocateBuffers(1, VK_COMMAND_BUFFER_LEVEL_PRIMARY);
	VkCommandBuffer commandBuffer = commandPool.GetBuffer(0);

	VkCommandBufferBeginInfo beginInfo;
	beginInfo.sType = VK_STRUCTURE_TYPE_COMMAND_BUFFER_BEGIN_INFO;
	beginInfo.pNext = nullptr;
	beginInfo.flags = 0;
	beginInfo.pInheritanceInfo = nullptr;

	result = vkBeginCommandBuffer(commandBuffer, &beginInfo);
	ASSERT_VULKAN(result);

	en::vk::CommandRecorder::ImageLayoutTransfer(
		commandBuffer,
		image,
		VK_IMAGE_LAYOUT_UNDEFINED,
		VK_IMAGE_LAYOUT_GENERAL,
		VK_ACCESS_NONE,
		VK_ACCESS_SHADER_WRITE_BIT | VK_ACCESS_SHADER_READ_BIT,
		VK_PIPELINE_STAGE_TOP_OF_PIPE_BIT,
		VK_PIPELINE_STAGE_COMPUTE_SHADER_BIT);

	result = vkEndCommandBuffer(commandBuffer);
	ASSERT_VULKAN(result);

	VkSubmitInfo submitInfo;
	submitInfo.sType = VK_STRUCTURE_TYPE_SUBMIT_INFO;
	submitInfo.pNext = nullptr;
	submitInfo.waitSemaphoreCount = 0;
	submitInfo.pWaitSemaphores = nullptr;
	submitInfo.pWaitDstStageMask = nullptr;
	submitInfo.commandBufferCount = 1;
	submitInfo.pCommandBuffers = &commandBuffer;
	submitInfo.signalSemaphoreCount = 0;
	submitInfo.pSignalSemaphores = nullptr;

	VkQueue queue = en::VulkanAPI::GetGraphicsQueue();
	result = vkQueueSubmit(queue, 1, &submitInfo, VK_NULL_HANDLE);
	ASSERT_VULKAN(result);
	result = vkQueueWaitIdle(queue);
	ASSERT_VULKAN(result);
}

HANDLE GetImageMemoryHandle()
{
	HANDLE handle;

	VkMemoryGetWin32HandleInfoKHR vkMemoryGetWin32HandleInfoKHR = {};
	vkMemoryGetWin32HandleInfoKHR.sType = VK_STRUCTURE_TYPE_MEMORY_GET_WIN32_HANDLE_INFO_KHR;
	vkMemoryGetWin32HandleInfoKHR.pNext = NULL;
	vkMemoryGetWin32HandleInfoKHR.memory = imageMemory;
	vkMemoryGetWin32HandleInfoKHR.handleType = (VkExternalMemoryHandleTypeFlagBitsKHR)VK_EXTERNAL_MEMORY_HANDLE_TYPE_OPAQUE_WIN32_BIT;

	fpGetMemoryWin32HandleKHR(en::VulkanAPI::GetDevice(), &vkMemoryGetWin32HandleInfoKHR, &handle);
	return handle;
}

void RunTcnn()
{
	// Start engine
	std::string appName("NRC-HPM-Renderer");
	uint32_t width = 768; // Multiple of 128 for nrc batch size
	uint32_t height = width;
	en::Log::Info("Starting " + appName);
	en::Window::Init(width, height, false, appName);
	en::VulkanAPI::Init(appName);

	// Init tcnn
	nlohmann::json config = {
	{"loss", {
		{"otype", "L2"}
	}},
	{"optimizer", {
		{"otype", "Adam"},
		{"learning_rate", 1e-3},
	}},
	{"encoding", {
		{"otype", "Composite"},
		{"reduction", "Concatenation"},
		{"nested", {
			{
				{"otype", "HashGrid"},
				{"n_dims_to_encode", 3},
				{"n_levels", 16},
				{"n_features_per_level", 2},
				{"log2_hashmap_size", 19},
				{"base_resolution", 16},
				{"per_level_scale", 2.0},
			},
			{
				{"otype", "OneBlob"},
				{"n_dims_to_encode", 2},
				{"n_bins", 4},
			},
		}},
	}},
	{"network", {
		{"otype", "FullyFusedMLP"},
		{"activation", "ReLU"},
		{"output_activation", "None"},
		{"n_neurons", 128},
		{"n_hidden_layers", 6},
	}},
	};

	const uint32_t n_input_dims = 5;
	const uint32_t n_output_dims = 3;
	const uint32_t n_inference_steps = 36;
	const uint32_t n_training_steps = 10;
	const uint32_t batch_size = 16384;

	tcnn::TrainableModel model = tcnn::create_from_config(n_input_dims, n_output_dims, config);

	tcnn::GPUMatrix<float> training_batch_inputs(n_input_dims, batch_size);
	tcnn::GPUMatrix<float> training_batch_targets(n_output_dims, batch_size);

	tcnn::GPUMatrix<float> inference_inputs(n_input_dims, batch_size);
	tcnn::GPUMatrix<float> inference_outputs(n_output_dims, batch_size);

	tcnn::GPUMemory<uint8_t> tcnnMemory(batch_size * n_input_dims * sizeof(float));

	// Interop test
	LoadVulkanProcAddr();
	CreateImage(width, height);
	
	hipExternalMemoryHandleDesc cuExtMemHandleDesc;
	memset(&cuExtMemHandleDesc, 0, sizeof(hipExternalMemoryHandleDesc));
	cuExtMemHandleDesc.type = hipExternalMemoryHandleTypeOpaqueWin32;
	cuExtMemHandleDesc.handle.win32.handle = GetImageMemoryHandle();
	cuExtMemHandleDesc.size = width * height * 4 * sizeof(float);
	
	hipExternalMemory_t cuVkImageMemory;
	hipError_t cudaResult = hipImportExternalMemory(&cuVkImageMemory, &cuExtMemHandleDesc);
	ASSERT_CUDA(cudaResult);

	// Main loop
	VkDevice device = en::VulkanAPI::GetDevice();
	VkQueue graphicsQueue = en::VulkanAPI::GetGraphicsQueue();
	VkResult result;
	while (!en::Window::IsClosed())
	{
		// Update
		en::Window::Update();
		width = en::Window::GetWidth();
		height = en::Window::GetHeight();
	}
	result = vkDeviceWaitIdle(device);
	ASSERT_VULKAN(result);

	// End
	en::VulkanAPI::Shutdown();
	en::Window::Shutdown();

	en::Log::Info("Ending " + appName);
}
