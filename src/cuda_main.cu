#include "hip/hip_runtime.h"
#include <engine/cuda_common.hpp>
#include <engine/util/Log.hpp>
#include <engine/graphics/Window.hpp>
#include <engine/graphics/VulkanAPI.hpp>
#include <engine/graphics/vulkan/CommandPool.hpp>
#include <engine/graphics/vulkan/CommandRecorder.hpp>
#include <engine/graphics/renderer/ImGuiRenderer.hpp>
#include <engine/graphics/vulkan/Swapchain.hpp>
#include <imgui.h>
#include <engine/graphics/renderer/NrcHpmRenderer.hpp>
#include <engine/graphics/NeuralRadianceCache.hpp>
#include <engine/util/read_file.hpp>
#include <engine/util/Input.hpp>
#include <engine/util/Time.hpp>
#include <engine/HpmScene.hpp>
#include <engine/AppConfig.hpp>
#include <engine/graphics/renderer/McHpmRenderer.hpp>
#include <engine/graphics/vulkan/CommandPool.hpp>
#include <engine/graphics/Reference.hpp>
#include <engine/objects/Model.hpp>
#include <engine/graphics/renderer/SimpleModelRenderer.hpp>
#include <openvdb/openvdb.h>
#include <engine/util/LogFile.hpp>

en::Reference* reference = nullptr;
en::NrcHpmRenderer* nrcHpmRenderer = nullptr;
en::McHpmRenderer* mcHpmRenderer = nullptr;

void RecordSwapchainCommandBuffer(VkCommandBuffer commandBuffer, VkImage image)
{
	uint32_t width = en::Window::GetWidth();
	uint32_t height = en::Window::GetHeight();

	VkCommandBufferBeginInfo beginInfo;
	beginInfo.sType = VK_STRUCTURE_TYPE_COMMAND_BUFFER_BEGIN_INFO;
	beginInfo.pNext = nullptr;
	beginInfo.flags = 0;
	beginInfo.pInheritanceInfo = nullptr;

	VkResult result = vkBeginCommandBuffer(commandBuffer, &beginInfo);
	if (result != VK_SUCCESS)
		en::Log::Error("Failed to begin VkCommandBuffer", true);

	en::vk::CommandRecorder::ImageLayoutTransfer(
		commandBuffer,
		image,
		VK_IMAGE_LAYOUT_UNDEFINED,
		VK_IMAGE_LAYOUT_TRANSFER_DST_OPTIMAL,
		VK_ACCESS_NONE_KHR,
		VK_ACCESS_TRANSFER_WRITE_BIT,
		VK_PIPELINE_STAGE_TOP_OF_PIPE_BIT,
		VK_PIPELINE_STAGE_TRANSFER_BIT);

	if (nrcHpmRenderer != nullptr && mcHpmRenderer != nullptr && en::ImGuiRenderer::IsInitialized())
	{
		VkImageCopy imageCopy;
		imageCopy.srcSubresource.aspectMask = VK_IMAGE_ASPECT_COLOR_BIT;
		imageCopy.srcSubresource.mipLevel = 0;
		imageCopy.srcSubresource.baseArrayLayer = 0;
		imageCopy.srcSubresource.layerCount = 1;
		imageCopy.srcOffset = { 0, 0, 0 };
		imageCopy.dstSubresource.aspectMask = VK_IMAGE_ASPECT_COLOR_BIT;
		imageCopy.dstSubresource.mipLevel = 0;
		imageCopy.dstSubresource.baseArrayLayer = 0;
		imageCopy.dstSubresource.layerCount = 1;
		imageCopy.dstOffset = { 0, 0, 0 };
		imageCopy.extent = { width, height, 1 };

		vkCmdCopyImage(
			commandBuffer,
			en::ImGuiRenderer::GetImage(),
			VK_IMAGE_LAYOUT_TRANSFER_SRC_OPTIMAL,
			image,
			VK_IMAGE_LAYOUT_TRANSFER_DST_OPTIMAL,
			1,
			&imageCopy);
	}

	en::vk::CommandRecorder::ImageLayoutTransfer(
		commandBuffer,
		image,
		VK_IMAGE_LAYOUT_TRANSFER_DST_OPTIMAL,
		VK_IMAGE_LAYOUT_PRESENT_SRC_KHR,
		VK_ACCESS_TRANSFER_WRITE_BIT,
		VK_ACCESS_COLOR_ATTACHMENT_READ_BIT,
		VK_PIPELINE_STAGE_TRANSFER_BIT,
		VK_PIPELINE_STAGE_COLOR_ATTACHMENT_OUTPUT_BIT);

	result = vkEndCommandBuffer(commandBuffer);
	if (result != VK_SUCCESS)
		en::Log::Error("Failed to end VkCommandBuffer", true);
}

void SwapchainResizeCallback()
{
	en::Window::WaitForUsableSize();
	vkDeviceWaitIdle(en::VulkanAPI::GetDevice());

	en::Log::Info("Skipping swapchain resize callback");

	//uint32_t width = en::Window::GetWidth();
	//uint32_t height = en::Window::GetHeight();
	//nrcHpmRenderer->ResizeFrame(width, height);
	//en::ImGuiRenderer::Resize(width, height);
	//en::ImGuiRenderer::SetBackgroundImageView(imageView);
}

struct ViewBenchmarkStats
{
	float mse;
	glm::vec3 bias;
};

struct BenchmarkStats
{
	size_t frameIndex;
	float frameTimeMS;
	float loss;
	std::array<ViewBenchmarkStats, 6> viewStats;

	std::string ToString() const
	{
		std::string str = 
			std::to_string(frameIndex) + " " +
			std::to_string(frameTimeMS) + " " +
			std::to_string(loss) + " ";
		for (size_t i = 0; i < viewStats.size(); i++)
		{
			str += std::to_string(viewStats[i].mse) + " " +
				std::to_string(viewStats[i].bias.x) + " " +
				std::to_string(viewStats[i].bias.y) + " " +
				std::to_string(viewStats[i].bias.z) + " ";
		}

		return str;
	}
};

void Benchmark(const en::Camera* camera, VkQueue queue, size_t frameCount, BenchmarkStats& stats, en::LogFile& logFile)
{
	en::Log::Info("Frame: " + std::to_string(frameCount));
	std::array<en::Reference::Result, 6> results = reference->CompareNrc(*nrcHpmRenderer, camera, queue);

	for (size_t i = 0; i < results.size(); i++)
	{
		stats.viewStats[i].mse = results[i].mse;
		stats.viewStats[i].bias.x = results[i].biasX;
		stats.viewStats[i].bias.y = results[i].biasY;
		stats.viewStats[i].bias.z = results[i].biasZ;
	}

	logFile.WriteLine(stats.ToString());
}

bool RunAppConfigInstance(const en::AppConfig& appConfig)
{
	// Start engine
	const std::string appName("NRC-HPM-Renderer");
	uint32_t width = 1920;
	uint32_t height = 1080;
	en::Log::Info("Starting " + appName);

	en::Window::Init(width, height, false, appName);
	if (en::Window::IsSupported()) { en::Input::Init(en::Window::GetGLFWHandle()); }
	en::VulkanAPI::Init(appName);
	const VkDevice device = en::VulkanAPI::GetDevice();
	const uint32_t qfi = en::VulkanAPI::GetGraphicsQFI();
	const VkQueue queue = en::VulkanAPI::GetGraphicsQueue();

	// Renderer select
	const std::vector<char*> rendererMenuItems = { "MC", "NRC", "Model" };
	const char* currentRendererMenuItem = rendererMenuItems[1];
	uint32_t rendererId = 1;

	// Init resources
	en::Log::Info("Initializing rendering resources");

	en::NeuralRadianceCache nrc(appConfig);

	en::HpmScene hpmScene(appConfig);

	const float aspectRatio = static_cast<float>(width) / static_cast<float>(height);
	en::Camera camera(
		glm::vec3(64.0f, 0.0f, 0.0f),
		glm::vec3(-1.0f, 0.0f, 0.0f),
		glm::vec3(0.0f, 1.0f, 0.0f),
		aspectRatio,
		glm::radians(60.0f),
		0.1f,
		100.0f);

	en::Model dragonModel("dragon.obj", true);
	en::ModelInstance dragonModelInstance(&dragonModel, glm::mat4(1.0f));

	// Init reference
	reference = new en::Reference(width, height, appConfig, hpmScene, queue);

	// Init rendering pipeline
	en::Log::Info("Initializing renderers");

	en::vk::Swapchain* swapchain = nullptr;
	if (en::Window::IsSupported())
	{
		swapchain = new en::vk::Swapchain(width, height, RecordSwapchainCommandBuffer, SwapchainResizeCallback);
	}

	en::SimpleModelRenderer modelRenderer(width, height, &camera);
	modelRenderer.AddModelInstance(&dragonModelInstance);

	nrcHpmRenderer = new en::NrcHpmRenderer(
		width,
		height,
		appConfig.trainSampleRatio,
		appConfig.trainSpp,
		appConfig.primaryRayLength,
		false,
		&camera,
		hpmScene,
		nrc);

	mcHpmRenderer = new en::McHpmRenderer(width, height, 32, false, &camera, hpmScene);

	if (en::Window::IsSupported())
	{
		en::ImGuiRenderer::Init(width, height);
		switch (rendererId)
		{
		case 0: // MC
			en::ImGuiRenderer::SetBackgroundImageView(mcHpmRenderer->GetImageView());
			break;
		case 1: // NRC
			en::ImGuiRenderer::SetBackgroundImageView(nrcHpmRenderer->GetImageView());
			break;
		case 2: // Model
			en::ImGuiRenderer::SetBackgroundImageView(modelRenderer.GetColorImageView());
			break;
		default: // Error
			en::Log::Error("Renderer ID is invalid", true);
			break;
		}
	}

	// Swapchain rerecording because imgui renderer is now available
	if (en::Window::IsSupported()) { swapchain->Resize(width, height); }

	// Main loop
	en::Log::Info("Starting main loop");
	BenchmarkStats stats;
	en::LogFile logFile("output/ " + appConfig.GetName() + "/log.txt");
	VkResult result;
	size_t frameCount = 0;
	bool shutdown = false;
	bool restartAfterClose = false;
	bool benchmark = true;
	bool continueLoop = en::Window::IsSupported() ? !en::Window::IsClosed() : true;
	while (continueLoop && !shutdown)
	{
		// Update
		if (en::Window::IsSupported())
		{
			en::Window::Update();
			en::Input::Update();
		}
		en::Time::Update();

		if (en::Window::IsSupported())
		{
			width = en::Window::GetWidth();
			height = en::Window::GetHeight();
		}

		float deltaTime = static_cast<float>(en::Time::GetDeltaTime());
		uint32_t fps = en::Time::GetFps();

		// Physics
		if (en::Window::IsSupported())
		{
			en::Input::HandleUserCamInput(&camera, deltaTime);
			camera.SetAspectRatio(width, height);
		}
		camera.UpdateUniformBuffer();

		// Render
		switch (rendererId)
		{
		case 0: // MC
			mcHpmRenderer->Render(queue);
			result = vkQueueWaitIdle(queue);
			ASSERT_VULKAN(result);
			mcHpmRenderer->EvaluateTimestampQueries();
			break;
		case 1: // NRC
			nrcHpmRenderer->Render(queue, true);
			result = vkQueueWaitIdle(queue);
			ASSERT_VULKAN(result);
			nrcHpmRenderer->EvaluateTimestampQueries();
			break;
		case 2: // Model
			modelRenderer.Render(queue);
			ASSERT_VULKAN(vkQueueWaitIdle(queue));
			break;
		default: // Error
			en::Log::Error("Renderer ID is invalid", true);
			break;
		}

		//
		const float nrcLoss = nrc.GetLoss();

		// Imgui
		if (en::Window::IsSupported())
		{
			en::ImGuiRenderer::StartFrame();

			ImGui::Begin("Statistics");
			ImGui::Text((std::string("Framecount ") + std::to_string(frameCount)).c_str());
			ImGui::Text("DeltaTime %f", deltaTime);
			ImGui::Text("FPS %d", fps);
			ImGui::Text("NRC Loss %f", nrcLoss);
			ImGui::End();

			ImGui::Begin("Controls");
			shutdown = ImGui::Button("Shutdown");
			ImGui::Checkbox("Restart after shutdown", &restartAfterClose);
			ImGui::Checkbox("Benchmark", &benchmark);

			if (ImGui::BeginCombo("##combo", currentRendererMenuItem))
			{
				for (int i = 0; i < rendererMenuItems.size(); i++)
				{
					bool selected = (currentRendererMenuItem == rendererMenuItems[i]);
					if (ImGui::Selectable(rendererMenuItems[i], selected))
					{
						if (i != rendererId)
						{
							rendererId = i;
							switch (rendererId)
							{
							case 0: // MC
								en::ImGuiRenderer::SetBackgroundImageView(mcHpmRenderer->GetImageView());
								break;
							case 1: // NRC
								en::ImGuiRenderer::SetBackgroundImageView(nrcHpmRenderer->GetImageView());
								break;
							case 2: // Model
								en::ImGuiRenderer::SetBackgroundImageView(modelRenderer.GetColorImageView());
								break;
							default: // Error
								en::Log::Error("Renderer ID is invalid", true);
								break;
							}
						}
						currentRendererMenuItem = rendererMenuItems[i];
					};
					if (selected) { ImGui::SetItemDefaultFocus(); }
				}
				ImGui::EndCombo();
			}

			ImGui::End();

			mcHpmRenderer->RenderImGui();
			nrcHpmRenderer->RenderImGui();

			hpmScene.Update(true);

			appConfig.RenderImGui();

			en::ImGuiRenderer::EndFrame(queue, VK_NULL_HANDLE);
			result = vkQueueWaitIdle(queue);
			ASSERT_VULKAN(result);
		}

		// Display
		if (en::Window::IsSupported()) { swapchain->DrawAndPresent(VK_NULL_HANDLE, VK_NULL_HANDLE); }

		// Benchmark
		stats.frameIndex = frameCount;
		stats.frameTimeMS = nrcHpmRenderer->GetFrameTimeMS();
		stats.loss = nrc.GetLoss();
		if (benchmark && frameCount % 1 == 0) { Benchmark(&camera, queue, frameCount, stats, logFile); }

		// Exit if loss is invalid
		if (std::isnan(nrcLoss) || std::isinf(nrcLoss))
		{
			en::Log::Error("NRC Loss is " + std::to_string(nrcLoss), false);
			break;
		}

		//
		frameCount++;
		continueLoop = en::Window::IsSupported() ? !en::Window::IsClosed() : true;
	}

	// Stop gpu work
	result = vkDeviceWaitIdle(device);
	ASSERT_VULKAN(result);

	// End
	mcHpmRenderer->Destroy();
	delete mcHpmRenderer;
	
	nrcHpmRenderer->Destroy();
	delete nrcHpmRenderer;
	en::ImGuiRenderer::Shutdown();
	if (en::Window::IsSupported) { swapchain->Destroy(true); }

	modelRenderer.Destroy();

	reference->Destroy();
	delete reference; 

	dragonModelInstance.Destroy();
	dragonModel.Destroy();
	camera.Destroy();
	hpmScene.Destroy();
	nrc.Destroy();

	en::VulkanAPI::Shutdown();
	if (en::Window::IsSupported()) { en::Window::Shutdown(); }
	en::Log::Info("Ending " + appName);

	return restartAfterClose;
}

int main(int argc, char** argv)
{
	// Init openvdb
	openvdb::initialize();

	// Read arguments for app config
	std::vector<char*> myargv(argc);
	std::memcpy(myargv.data(), argv, sizeof(char*) * argc);
	if (argc == 1)
	{
		en::Log::Info("No arguments found. Loading defaults");
		myargv = { 
			"NRC-HPM-Renderer", 
			"RelativeL2", "Adam", "0.001", "0.99",
			"2", "0", 
			"64", "6", "15", 
			"1", 
			"0.05", "1.0", "1", "2"
		};
	}

	// Create app config
	en::AppConfig appConfig(myargv);

	// Create output path if not exists
	std::string outputDirPath = "output/ " + appConfig.GetName() + "/";
	if (!std::filesystem::is_directory(outputDirPath) || !std::filesystem::exists(outputDirPath))
	{
		std::filesystem::create_directory(outputDirPath);
	}

	// Run
	bool restartRunConfig;
	do {
		restartRunConfig = RunAppConfigInstance(appConfig);
	} while (restartRunConfig);

	// Exit
	return 0;
}
