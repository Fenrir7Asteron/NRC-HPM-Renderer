#include "hip/hip_runtime.h"
#include <engine/cuda_common.hpp>
#include <engine/util/Log.hpp>
#include <engine/graphics/Window.hpp>
#include <engine/graphics/VulkanAPI.hpp>
#include <engine/graphics/vulkan/CommandPool.hpp>
#include <engine/graphics/vulkan/CommandRecorder.hpp>
#include <engine/graphics/renderer/ImGuiRenderer.hpp>
#include <engine/graphics/vulkan/Swapchain.hpp>
#include <imgui.h>
#include <engine/graphics/renderer/NrcHpmRenderer.hpp>
#include <engine/graphics/NeuralRadianceCache.hpp>
#include <engine/util/read_file.hpp>
#include <engine/util/Input.hpp>
#include <engine/util/Time.hpp>
#include <engine/HpmScene.hpp>
#include <engine/AppConfig.hpp>
#include <engine/graphics/renderer/McHpmRenderer.hpp>
#include <engine/graphics/vulkan/CommandPool.hpp>
#include <engine/graphics/Reference.hpp>
#include <engine/objects/Model.hpp>
#include <engine/graphics/renderer/SimpleModelRenderer.hpp>
#include <engine/util/LogFile.hpp>
#include <openvdb/openvdb.h>
#include <filesystem>

en::Reference* reference = nullptr;
en::NrcHpmRenderer* nrcHpmRenderer = nullptr;
en::McHpmRenderer* mcHpmRenderer = nullptr;
bool renderGui = true;

void RecordSwapchainCommandBuffer(VkCommandBuffer commandBuffer, VkImage image)
{
	uint32_t width = en::Window::GetWidth();
	uint32_t height = en::Window::GetHeight();

	VkCommandBufferBeginInfo beginInfo;
	beginInfo.sType = VK_STRUCTURE_TYPE_COMMAND_BUFFER_BEGIN_INFO;
	beginInfo.pNext = nullptr;
	beginInfo.flags = 0;
	beginInfo.pInheritanceInfo = nullptr;

	VkResult result = vkBeginCommandBuffer(commandBuffer, &beginInfo);
	if (result != VK_SUCCESS)
		en::Log::Error("Failed to begin VkCommandBuffer", true);

	en::vk::CommandRecorder::ImageLayoutTransfer(
		commandBuffer,
		image,
		VK_IMAGE_LAYOUT_UNDEFINED,
		VK_IMAGE_LAYOUT_TRANSFER_DST_OPTIMAL,
		VK_ACCESS_NONE_KHR,
		VK_ACCESS_TRANSFER_WRITE_BIT,
		VK_PIPELINE_STAGE_TOP_OF_PIPE_BIT,
		VK_PIPELINE_STAGE_TRANSFER_BIT);

	if (nrcHpmRenderer != nullptr && mcHpmRenderer != nullptr && en::ImGuiRenderer::IsInitialized())
	{
		VkImageCopy imageCopy;
		imageCopy.srcSubresource.aspectMask = VK_IMAGE_ASPECT_COLOR_BIT;
		imageCopy.srcSubresource.mipLevel = 0;
		imageCopy.srcSubresource.baseArrayLayer = 0;
		imageCopy.srcSubresource.layerCount = 1;
		imageCopy.srcOffset = { 0, 0, 0 };
		imageCopy.dstSubresource.aspectMask = VK_IMAGE_ASPECT_COLOR_BIT;
		imageCopy.dstSubresource.mipLevel = 0;
		imageCopy.dstSubresource.baseArrayLayer = 0;
		imageCopy.dstSubresource.layerCount = 1;
		imageCopy.dstOffset = { 0, 0, 0 };
		imageCopy.extent = { width, height, 1 };

		vkCmdCopyImage(
			commandBuffer,
			en::ImGuiRenderer::GetImage(),
			VK_IMAGE_LAYOUT_TRANSFER_SRC_OPTIMAL,
			image,
			VK_IMAGE_LAYOUT_TRANSFER_DST_OPTIMAL,
			1,
			&imageCopy);
	}

	en::vk::CommandRecorder::ImageLayoutTransfer(
		commandBuffer,
		image,
		VK_IMAGE_LAYOUT_TRANSFER_DST_OPTIMAL,
		VK_IMAGE_LAYOUT_PRESENT_SRC_KHR,
		VK_ACCESS_TRANSFER_WRITE_BIT,
		VK_ACCESS_COLOR_ATTACHMENT_READ_BIT,
		VK_PIPELINE_STAGE_TRANSFER_BIT,
		VK_PIPELINE_STAGE_COLOR_ATTACHMENT_OUTPUT_BIT);

	result = vkEndCommandBuffer(commandBuffer);
	if (result != VK_SUCCESS)
		en::Log::Error("Failed to end VkCommandBuffer", true);
}

void SwapchainResizeCallback()
{
	en::Window::WaitForUsableSize();
	vkDeviceWaitIdle(en::VulkanAPI::GetDevice());

	en::Log::Info("Skipping swapchain resize callback");

	//uint32_t width = en::Window::GetWidth();
	//uint32_t height = en::Window::GetHeight();
	//nrcHpmRenderer->ResizeFrame(width, height);
	//en::ImGuiRenderer::Resize(width, height);
	//en::ImGuiRenderer::SetBackgroundImageView(imageView);
}

struct ViewBenchmarkStats
{
	float mse;
	glm::vec3 bias;
};

struct BenchmarkStats
{
	size_t frameIndex;
	float frameTimeMS;
	float loss;
	std::array<ViewBenchmarkStats, 6> viewStats;

	std::string ToString() const
	{
		std::string str = 
			std::to_string(frameIndex) + " " +
			std::to_string(frameTimeMS) + " " +
			std::to_string(loss) + " ";
		for (size_t i = 0; i < viewStats.size(); i++)
		{
			str += std::to_string(viewStats[i].mse) + " " +
				std::to_string(viewStats[i].bias.x) + " " +
				std::to_string(viewStats[i].bias.y) + " " +
				std::to_string(viewStats[i].bias.z) + " ";
		}

		return str;
	}
};

void Benchmark(const en::Camera* camera, VkQueue queue, size_t frameCount, BenchmarkStats& stats, en::LogFile& logFileNrc, en::LogFile& logFileMc)
{
	en::Log::Info("Frame: " + std::to_string(frameCount));
	en::Reference::Result nrcResult = reference->CompareNrc(*nrcHpmRenderer, camera, queue);
	en::Reference::Result mcResult = reference->CompareMc(*mcHpmRenderer, camera, queue);
	logFileNrc.WriteLine(
		std::to_string(frameCount) + " " + 
		std::to_string(nrcResult.mse) + " " +
		std::to_string(nrcResult.GetRelBias()) + " " +
		std::to_string(nrcResult.GetCV()) + " " + 
		std::to_string(nrcHpmRenderer->GetLoss())
	);

	logFileMc.WriteLine(
		std::to_string(frameCount) + " " +
		std::to_string(mcResult.mse) + " " +
		std::to_string(mcResult.GetRelBias()) + " " +
		std::to_string(mcResult.GetCV())
	);
}

std::string GetCurrentTimestampString()
{
	auto t = std::time(nullptr);
	auto tm = *std::localtime(&t);

	std::ostringstream oss;
	oss << std::put_time(&tm, "(%d-%m-%Y_%H-%M-%S)");
	return oss.str();
}

void CreateOutputDirectory(std::string& outputDirPath)
{
	// Create output path if not exists
	if (!std::filesystem::is_directory(outputDirPath) || !std::filesystem::exists(outputDirPath))
	{
		std::filesystem::create_directories(outputDirPath);
	}
}

bool RunAppConfigInstance(const en::AppConfig& appConfig)
{
	// Start engine
	const std::string appName("NRC-HPM-Renderer");
	uint32_t width = 1920;
	uint32_t height = 1080;
	en::Log::Info("Starting " + appName);

	en::Window::Init(width, height, false, appName);
	if (en::Window::IsSupported()) { en::Input::Init(en::Window::GetGLFWHandle()); }
	en::VulkanAPI::Init(appName);
	const VkDevice device = en::VulkanAPI::GetDevice();
	const uint32_t qfi = en::VulkanAPI::GetGraphicsQFI();
	const VkQueue queue = en::VulkanAPI::GetGraphicsQueue();

	// Renderer select
	const std::vector<char*> rendererMenuItems = { "MC", "NRC", "Model" };
	const char* currentRendererMenuItem = rendererMenuItems[1];
	uint32_t rendererId = 1;

	// Init resources
	en::Log::Info("Initializing rendering resources");

	en::NeuralRadianceCache nrc(appConfig, width, height);

	en::HpmScene hpmScene(appConfig);

	const float aspectRatio = static_cast<float>(width) / static_cast<float>(height);
	en::Camera camera(
		glm::vec3(64.0f, 0.0f, 0.0f),
		glm::vec3(-1.0f, 0.0f, 0.0f),
		glm::vec3(0.0f, 1.0f, 0.0f),
		aspectRatio,
		glm::radians(60.0f),
		0.1f,
		100.0f);

	// Init reference
	if (!hpmScene.IsDynamic()) { reference = new en::Reference(width, height, appConfig, hpmScene, queue); }

	// Init rendering pipeline
	en::Log::Info("Initializing renderers");

	en::vk::Swapchain* swapchain = nullptr;
	if (en::Window::IsSupported())
	{
		swapchain = new en::vk::Swapchain(width, height, RecordSwapchainCommandBuffer, SwapchainResizeCallback);
	}

	en::SimpleModelRenderer modelRenderer(width, height, &camera);
	
	nrcHpmRenderer = new en::NrcHpmRenderer(
		width,
		height,
		false,
		&camera,
		appConfig,
		hpmScene,
		nrc);

	mcHpmRenderer = new en::McHpmRenderer(width, height, 32, false, &camera, hpmScene);

	if (en::Window::IsSupported())
	{
		en::ImGuiRenderer::Init(width, height);
		switch (rendererId)
		{
		case 0: // MC
			en::ImGuiRenderer::SetBackgroundImageView(mcHpmRenderer->GetImageView());
			break;
		case 1: // NRC
			en::ImGuiRenderer::SetBackgroundImageView(nrcHpmRenderer->GetImageView());
			break;
		case 2: // Model
			en::ImGuiRenderer::SetBackgroundImageView(modelRenderer.GetColorImageView());
			break;
		default: // Error
			en::Log::Error("Renderer ID is invalid", true);
			break;
		}
	}

	// Swapchain rerecording because imgui renderer is now available
	if (en::Window::IsSupported()) { swapchain->Resize(width, height); }

	// Main loop
	en::Log::Info("Starting main loop");
	BenchmarkStats stats;
	std::string outputDirPath = "output/" + appConfig.GetName() + GetCurrentTimestampString() + "/";
	en::LogFile logFileNrc(outputDirPath + "/logNrc.txt");
	en::LogFile logFileMc(outputDirPath + "/logMc.txt");
	VkResult result;
	size_t frameCount = 0;
	bool shutdown = false;
	bool restartAfterClose = false;

	bool benchmark = appConfig.enableBenchmarkOnStart;
	if (benchmark)
	{
		CreateOutputDirectory(outputDirPath);
	}

	bool continueLoop = en::Window::IsSupported() ? !en::Window::IsClosed() : true;
	bool pause = appConfig.enablePauseOnStart;
	bool pauseAfterNFrames = 0; // if N > 0, then set pause = true after N frames

	while (continueLoop && !shutdown)
	{
		// Update
		if (en::Window::IsSupported())
		{
			en::Window::Update();
			en::Input::Update();
		}
		en::Time::Update();

		if (en::Window::IsSupported())
		{
			width = en::Window::GetWidth();
			height = en::Window::GetHeight();
		}

		float deltaTime = static_cast<float>(en::Time::GetDeltaTime());
		uint32_t fps = en::Time::GetFps();

		// Physics
		if (en::Window::IsSupported())
		{
			en::Input::HandleUserCamInput(&camera, deltaTime);
			camera.SetAspectRatio(width, height);
		}
		camera.UpdateUniformBuffer();

		// Render
		if (!pause)
		{
			switch (rendererId)
			{
			case 0: // MC
				mcHpmRenderer->Render(queue);
				result = vkQueueWaitIdle(queue);
				ASSERT_VULKAN(result);
				mcHpmRenderer->EvaluateTimestampQueries();
				break;
			case 1: // NRC
				nrcHpmRenderer->Render(queue, true);
				result = vkQueueWaitIdle(queue);
				ASSERT_VULKAN(result);
				nrcHpmRenderer->EvaluateTimestampQueries();
				break;
			case 2: // Model
				modelRenderer.Render(queue);
				ASSERT_VULKAN(vkQueueWaitIdle(queue));
				break;
			default: // Error
				en::Log::Error("Renderer ID is invalid", true);
				break;
			}
		}

		//
		const float nrcLoss = nrc.GetLoss();

		// Imgui
		if (en::Input::IsKeyPressed(en::KEY_H))
		{
			renderGui = false;
			en::Log::Info("RenderGUI enabled: false");
		}

		if (en::Input::IsKeyPressed(en::KEY_J))
		{
			renderGui = true;
			en::Log::Info("RenderGUI enabled: true");
		}

		if (!pause && pauseAfterNFrames > 0)
		{
			pauseAfterNFrames -= 1;
			if (pauseAfterNFrames <= 0)
			{
				pause = true;
			}
		}

		if (en::Window::IsSupported())
		{
			if (renderGui)
			{
				en::ImGuiRenderer::StartFrame();

				ImGui::Begin("Statistics");
				ImGui::Text((std::string("Framecount ") + std::to_string(frameCount)).c_str());
				ImGui::Text("DeltaTime %f", deltaTime);
				ImGui::Text("FPS %d", fps);
				ImGui::Text("NRC Loss %f", nrcLoss);
				ImGui::End();

				ImGui::Begin("Controls");
				shutdown = ImGui::Button("Shutdown");
				ImGui::Checkbox("Restart after shutdown", &restartAfterClose);

				bool benchmarkPreviousValue = benchmark;
				ImGui::Checkbox("Benchmark", &benchmark);
				if (benchmark && !benchmarkPreviousValue) // on enable
				{
					CreateOutputDirectory(outputDirPath);
				}

				ImGui::Checkbox("Pause", &pause);
				
				bool pauseAfterNFramesWasZero = pauseAfterNFrames == 0;
				ImGui::Checkbox("Advance one frame", &pauseAfterNFrames);
				if (pauseAfterNFrames > 0 && pauseAfterNFramesWasZero)
				{
					pause = false;
				}

				if (ImGui::BeginCombo("##combo", currentRendererMenuItem))
				{
					for (int i = 0; i < rendererMenuItems.size(); i++)
					{
						bool selected = (currentRendererMenuItem == rendererMenuItems[i]);
						if (ImGui::Selectable(rendererMenuItems[i], selected))
						{
							if (i != rendererId)
							{
								rendererId = i;
								switch (rendererId)
								{
								case 0: // MC
									en::ImGuiRenderer::SetBackgroundImageView(mcHpmRenderer->GetImageView());
									break;
								case 1: // NRC
									en::ImGuiRenderer::SetBackgroundImageView(nrcHpmRenderer->GetImageView());
									break;
								case 2: // Model
									en::ImGuiRenderer::SetBackgroundImageView(modelRenderer.GetColorImageView());
									break;
								default: // Error
									en::Log::Error("Renderer ID is invalid", true);
									break;
								}
							}
							currentRendererMenuItem = rendererMenuItems[i];
						};
						if (selected) { ImGui::SetItemDefaultFocus(); }
					}
					ImGui::EndCombo();
				}

				ImGui::End();

				mcHpmRenderer->RenderImGui();
				nrcHpmRenderer->RenderImGui();

				hpmScene.RenderImGui();

				appConfig.RenderImGui();

				en::ImGuiRenderer::EndFrame(queue, VK_NULL_HANDLE);
				result = vkQueueWaitIdle(queue);
				ASSERT_VULKAN(result);
			}
			else
			{
				en::ImGuiRenderer::StartFrame();

				switch (rendererId)
				{
				case 0: // MC
					en::ImGuiRenderer::SetBackgroundImageView(mcHpmRenderer->GetImageView());
					break;
				case 1: // NRC
					en::ImGuiRenderer::SetBackgroundImageView(nrcHpmRenderer->GetImageView());
					break;
				case 2: // Model
					en::ImGuiRenderer::SetBackgroundImageView(modelRenderer.GetColorImageView());
					break;
				default: // Error
					en::Log::Error("Renderer ID is invalid", true);
					break;
				}

				en::ImGuiRenderer::EndFrame(queue, VK_NULL_HANDLE);
				result = vkQueueWaitIdle(queue);
				ASSERT_VULKAN(result);
			}
		}

		// Update scene
		hpmScene.Update(deltaTime);

		// Display
		if (en::Window::IsSupported()) { swapchain->DrawAndPresent(VK_NULL_HANDLE, VK_NULL_HANDLE); }

		// Benchmark
		stats.frameIndex = frameCount;
		stats.frameTimeMS = nrcHpmRenderer->GetFrameTimeMS();
		stats.loss = nrc.GetLoss();
		if (benchmark && !hpmScene.IsDynamic() && frameCount % 1 == 0) { Benchmark(&camera, queue, frameCount, stats, logFileNrc, logFileMc); }

		// Exit if loss is invalid
		if (std::isnan(nrcLoss) || std::isinf(nrcLoss))
		{
			en::Log::Error("NRC Loss is " + std::to_string(nrcLoss), false);
			break;
		}

		// Exit

		//
		if (!pause)
			frameCount++;

		continueLoop = en::Window::IsSupported() ? !en::Window::IsClosed() : true;
	}

	// Stop gpu work
	result = vkDeviceWaitIdle(device);
	ASSERT_VULKAN(result);

	// End
	mcHpmRenderer->Destroy();
	delete mcHpmRenderer;
	
	nrcHpmRenderer->Destroy();
	delete nrcHpmRenderer;
	en::ImGuiRenderer::Shutdown();
	if (en::Window::IsSupported) { swapchain->Destroy(true); }

	modelRenderer.Destroy();

	if (!hpmScene.IsDynamic()) { reference->Destroy(); delete reference; }

	camera.Destroy();
	hpmScene.Destroy();
	nrc.Destroy();

	en::VulkanAPI::Shutdown();
	if (en::Window::IsSupported()) { en::Window::Shutdown(); }
	en::Log::Info("Ending " + appName);

	return restartAfterClose;
}

int main(int argc, char** argv)
{
	// Init openvdb
	openvdb::initialize();

	// Read arguments for app config
	std::vector<char*> myargv(argc);
	std::memcpy(myargv.data(), argv, sizeof(char*) * argc);
	if (argc == 1)
	{
		en::Log::Info("No arguments found. Loading defaults");
		myargv = { 
			"NRC-HPM-Renderer", 
			"RelativeL2Luminance", "Adam", "0.01", "0.99",
			"0", "0", 
			"64", "6", "18", "14", "2", "2",
			"4", 
			"1.0", "1", "1", "0.0", "32",
			"0", "0"
		};
	}

	// Create app config
	en::AppConfig appConfig(myargv);

	// Run
	bool restartRunConfig;
	do {
		restartRunConfig = RunAppConfigInstance(appConfig);
	} while (restartRunConfig);

	// Exit
	return 0;
}
