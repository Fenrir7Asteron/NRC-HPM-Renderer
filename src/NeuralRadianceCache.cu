#include "hip/hip_runtime.h"
#include <engine/cuda_common.hpp>
#include <engine/graphics/NeuralRadianceCache.hpp>
#include <random>
#include <engine/util/Log.hpp>

namespace en
{
	uint32_t NeuralRadianceCache::sc_InputCount = 5;
	uint32_t NeuralRadianceCache::sc_OutputCount = 3;

	NeuralRadianceCache::NeuralRadianceCache(const AppConfig& appConfig) :
		m_InferBatchSize(2 << (appConfig.log2InferBatchSize - 1)),
		m_TrainBatchSize(2 << (appConfig.log2TrainBatchSize - 1)),
		m_TrainBatchCount(appConfig.trainBatchCount)
	{
		nlohmann::json modelConfig = {
			{"loss", {
				{"otype", appConfig.lossFn}
			}},
			{"optimizer", {
				{"otype", "EMA"},
				{"decay", appConfig.emaDecay},
				{"nested", {
					{"otype", appConfig.optimizer},
					{"learning_rate", appConfig.learningRate},
					//{"l2_reg", 0.0001},
				}}
			}},
			appConfig.encoding.jsonConfig,
			{"network", {
				{"otype", "FullyFusedMLP"},
				{"activation", "ReLU"},
				{"output_activation", "None"},
				{"n_neurons", appConfig.nnWidth},
				{"n_hidden_layers", appConfig.nnDepth},
			}},
		};

		m_Model = tcnn::create_from_config(sc_InputCount, sc_OutputCount, modelConfig);
	}

	void NeuralRadianceCache::Init(
		uint32_t inferCount,
		float* dCuInferInput,
		float* dCuInferOutput,
		float* dCuTrainInput,
		float* dCuTrainTarget,
		hipExternalSemaphore_t cudaStartSemaphore,
		hipExternalSemaphore_t cudaFinishedSemaphore)
	{
		// Check if sample counts are compatible
		if (inferCount % 16 != 0) { en::Log::Error("NRC requires inferCount to be a multiple of 16", true); }

		// Init members
		m_CudaStartSemaphore = cudaStartSemaphore;
		m_CudaFinishedSemaphore = cudaFinishedSemaphore;

		// Init big buffer
		const uint32_t trainCount = m_TrainBatchCount * m_TrainBatchSize;

		m_InferInput = tcnn::GPUMatrix<float>(dCuInferInput, sc_InputCount, inferCount);
		m_InferOutput = tcnn::GPUMatrix<float>(dCuInferOutput, sc_OutputCount, inferCount);
		m_TrainInput = tcnn::GPUMatrix<float>(dCuTrainInput, sc_InputCount, trainCount);
		m_TrainTarget = tcnn::GPUMatrix<float>(dCuTrainTarget, sc_OutputCount, trainCount);

		// Init infer buffers
		const uint32_t inferBatchCount = inferCount / m_InferBatchSize;
		const uint32_t inferLastBatchSize = inferCount - (inferBatchCount * m_InferBatchSize);
		m_InferInputBatches.resize(inferBatchCount);
		m_InferOutputBatches.resize(inferBatchCount);
		
		for (uint32_t i = 0; i < inferBatchCount; i++)
		{
			m_InferInputBatches[i] = m_InferInput.slice_cols(i * m_InferBatchSize, m_InferBatchSize);
			m_InferOutputBatches[i] = m_InferOutput.slice_cols(i * m_InferBatchSize, m_InferBatchSize);
		}

		if (inferLastBatchSize > 0)
		{
			m_InferInputBatches.push_back(m_InferInput.slice_cols(inferBatchCount * m_InferBatchSize, inferLastBatchSize));
			m_InferOutputBatches.push_back(m_InferOutput.slice_cols(inferBatchCount * m_InferBatchSize, inferLastBatchSize));
		}

		// Init train buffers
		m_TrainInputBatches.resize(m_TrainBatchCount);
		m_TrainTargetBatches.resize(m_TrainBatchCount);

		for (uint32_t i = 0; i < m_TrainBatchCount; i++)
		{
			m_TrainInputBatches[i] = m_TrainInput.slice_cols(i * m_TrainBatchSize, m_TrainBatchSize);
			m_TrainTargetBatches[i] = m_TrainTarget.slice_cols(i * m_TrainBatchSize, m_TrainBatchSize);
		}

		en::Log::Info("Infer batch count: " + std::to_string(inferBatchCount));
		en::Log::Info("Train batch count: " + std::to_string(m_TrainBatchCount));
	}

	void NeuralRadianceCache::InferAndTrain(const uint32_t* inferFilter, const uint32_t* trainFilter, uint32_t* trainFilteredFrameCounter, bool train)
	{
		AwaitCudaStartSemaphore();
		Inference(inferFilter);
		if (train) { Train(trainFilter, trainFilteredFrameCounter); }
		SignalCudaFinishedSemaphore();
	}

	void NeuralRadianceCache::Destroy()
	{
	}

	float NeuralRadianceCache::GetLoss() const
	{
		return m_Loss;
	}

	size_t NeuralRadianceCache::GetInferBatchCount() const
	{
		return m_InferInputBatches.size();
	}

	size_t NeuralRadianceCache::GetTrainBatchCount() const
	{
		return m_TrainInputBatches.size();
	}

	uint32_t NeuralRadianceCache::GetInferBatchSize() const
	{
		return m_InferBatchSize;
	}

	uint32_t NeuralRadianceCache::GetTrainBatchSize() const
	{
		return m_TrainBatchSize;
	}

	void NeuralRadianceCache::Inference(const uint32_t* inferFilter)
	{
		for (size_t i = 0; i < m_InferInputBatches.size(); i++)
		{
			if (inferFilter[i] > 0)
			{
				const tcnn::GPUMatrix<float>& inputBatch = m_InferInputBatches[i];
				tcnn::GPUMatrix<float>& outputBatch = m_InferOutputBatches[i];
				m_Model.network->inference(inputBatch, outputBatch);
			}
		}
	}

	void NeuralRadianceCache::Train(const uint32_t* trainFilter, uint32_t* trainFilteredFrameCounter)
	{
		for (size_t i = 0; i < m_TrainInputBatches.size(); i++)
		{
			if (trainFilter[i] <= 0)
			{
				trainFilteredFrameCounter[i] = std::min(trainFilteredFrameCounter[i] + 1, sc_FilterFrameCountThreshold);
			}
			else
			{
				trainFilteredFrameCounter[i] = 0;
			}

			// Exclude batch from training if it filtered more than sc_FilterFrameCountThreshold times
			// Batch is filtered if not a single ray scattered inside it
			if (trainFilteredFrameCounter[i] < sc_FilterFrameCountThreshold)
			{
				const tcnn::GPUMatrix<float>& inputBatch = m_TrainInputBatches[i];
				const tcnn::GPUMatrix<float>& targetBatch = m_TrainTargetBatches[i];
				auto forwardContext = m_Model.trainer->training_step(inputBatch, targetBatch);
				m_Loss = m_Model.trainer->loss(*forwardContext.get());
			}
		}
	}

	void NeuralRadianceCache::AwaitCudaStartSemaphore()
	{
		hipExternalSemaphoreWaitParams extSemaphoreWaitParams;
		memset(&extSemaphoreWaitParams, 0, sizeof(extSemaphoreWaitParams));
		extSemaphoreWaitParams.params.fence.value = 0;
		extSemaphoreWaitParams.flags = 0;

		hipError_t error = hipWaitExternalSemaphoresAsync(&m_CudaStartSemaphore, &extSemaphoreWaitParams, 1);
		ASSERT_CUDA(error);
	}

	void NeuralRadianceCache::SignalCudaFinishedSemaphore()
	{
		hipExternalSemaphoreSignalParams extSemaphoreSignalParams;
		memset(&extSemaphoreSignalParams, 0, sizeof(extSemaphoreSignalParams));
		extSemaphoreSignalParams.params.fence.value = 0;
		extSemaphoreSignalParams.flags = 0;

		hipError_t error = hipSignalExternalSemaphoresAsync(&m_CudaFinishedSemaphore, &extSemaphoreSignalParams, 1);
		ASSERT_CUDA(error);
	}
}
