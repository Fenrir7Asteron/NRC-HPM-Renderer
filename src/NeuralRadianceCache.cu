#include "hip/hip_runtime.h"
#include <engine/cuda_common.hpp>
#include <engine/graphics/NeuralRadianceCache.hpp>
#include <random>
#include <engine/util/Log.hpp>
#include <__msvc_chrono.hpp>

namespace en
{
	uint32_t NeuralRadianceCache::sc_InputCount = 6;
	uint32_t NeuralRadianceCache::sc_OutputCount = 3;

	NeuralRadianceCache::NeuralRadianceCache(const AppConfig& appConfig, const uint32_t renderWidth, const uint32_t renderHeight) :
		m_TrainBatchCountVertical(1 << (appConfig.maxTrainBatchLevel)),
		m_TrainBatchCountHorizontal(1 << (appConfig.maxTrainBatchLevel)),
		m_TrainMaxBatchLevel(appConfig.maxTrainBatchLevel),
		m_InferBatchSize(2 << (appConfig.log2InferBatchSize - 1)),
		m_TrainBatchSize(2 << (appConfig.log2TrainBatchSize - 1)),
		m_InferBatchSizeVertical(sqrt(2 << (appConfig.log2InferBatchSize - 1))),
		m_InferBatchSizeHorizontal(sqrt(2 << (appConfig.log2InferBatchSize - 1))),
		m_InferBatchCountVertical(ceil((float) renderHeight / m_InferBatchSizeVertical)),
		m_InferBatchCountHorizontal(ceil((float) renderWidth / m_InferBatchSizeHorizontal)),
		m_TrainBatchSizeVertical(sqrt(2 << (appConfig.log2TrainBatchSize - 1))),
		m_TrainBatchSizeHorizontal(sqrt(2 << (appConfig.log2TrainBatchSize - 1)))
	{
		nlohmann::json modelConfig = {
			{"loss", {
				{"otype", appConfig.lossFn}
			}},
			{"optimizer", {
				{"otype", "EMA"},
				{"decay", appConfig.emaDecay},
				{"nested", {
					{"otype", appConfig.optimizer},
					{"learning_rate", appConfig.learningRate},
					//{"l2_reg", 0.0001},
				}}
			}},
			appConfig.encoding.jsonConfig,
			{"network", {
				{"otype", "FullyFusedMLP"},
				{"activation", "ReLU"},
				{"output_activation", "None"},
				{"n_neurons", appConfig.nnWidth},
				{"n_hidden_layers", appConfig.nnDepth},
			}},
		};

		m_Model = tcnn::create_from_config(sc_InputCount, sc_OutputCount, modelConfig);
	}

	void NeuralRadianceCache::Init(
		float renderWidth, 
		float renderHeight,
		float* dCuInferInput,
		float* dCuInferOutput,
		float* dCuTrainInput,
		float* dCuTrainTarget,
		hipExternalSemaphore_t cudaStartSemaphore,
		hipExternalSemaphore_t cudaFinishedSemaphore)
	{
		const uint32_t inferCount = renderWidth * renderHeight;

		// Check if sample counts are compatible
		if (inferCount % 16 != 0) { en::Log::Error("NRC requires inferCount to be a multiple of 16", true); }

		// Init members
		m_CudaStartSemaphore = cudaStartSemaphore;
		m_CudaFinishedSemaphore = cudaFinishedSemaphore;

		// Init big buffer
		const uint32_t trainCount = m_TrainBatchCountVertical * m_TrainBatchCountHorizontal * m_TrainBatchSizeVertical * m_TrainBatchSizeHorizontal;

		m_InferInput = tcnn::GPUMatrix<float>(dCuInferInput, sc_InputCount, inferCount);
		m_InferOutput = tcnn::GPUMatrix<float>(dCuInferOutput, sc_OutputCount, inferCount);
		m_TrainInput = tcnn::GPUMatrix<float>(dCuTrainInput, sc_InputCount, trainCount);
		m_TrainTarget = tcnn::GPUMatrix<float>(dCuTrainTarget, sc_OutputCount, trainCount);

		// Init infer buffers
		
		uint32_t inferBatchCount = m_InferBatchCountVertical * m_InferBatchCountHorizontal;
		if (m_InferBatchSize % tcnn::BATCH_SIZE_GRANULARITY != 0) { en::Log::Error("NRC requires inferBatchSize to be a multiple of " + std::to_string(tcnn::BATCH_SIZE_GRANULARITY), true); }
		m_InferInputBatches.resize(inferBatchCount);
		m_InferOutputBatches.resize(inferBatchCount);

		const uint32_t inferLastBatchSizeVertical = renderHeight - ((m_InferBatchCountVertical - 1) * m_InferBatchSizeVertical);
		const uint32_t inferLastBatchSizeHorizontal = renderWidth - ((m_InferBatchCountHorizontal - 1) * m_InferBatchSizeHorizontal);
		uint32_t batchOffset = 0;

		for (uint32_t i = 0; i < m_InferBatchCountVertical; i++)
		{
			for (uint32_t j = 0; j < m_InferBatchCountHorizontal; j++)
			{
				const uint32_t linearBatchIdx = GetLinearInferBatchIndex(i, j);
				if (i < m_InferBatchCountVertical - 1 && j < m_InferBatchCountHorizontal - 1)
				{
					m_InferInputBatches[linearBatchIdx] = m_InferInput.slice_cols(batchOffset, m_InferBatchSize);
					m_InferOutputBatches[linearBatchIdx] = m_InferOutput.slice_cols(batchOffset, m_InferBatchSize);
					batchOffset += m_InferBatchSize;
				}
				else if (i == m_InferBatchCountVertical - 1 && j < m_InferBatchCountHorizontal - 1)
				{
					m_InferInputBatches[linearBatchIdx] = m_InferInput.slice_cols(batchOffset, inferLastBatchSizeVertical * m_InferBatchSizeHorizontal);
					m_InferOutputBatches[linearBatchIdx] = m_InferOutput.slice_cols(batchOffset, inferLastBatchSizeVertical * m_InferBatchSizeHorizontal);
					batchOffset += inferLastBatchSizeVertical * m_InferBatchSizeHorizontal;
				}
				else if (i < m_InferBatchCountVertical - 1 && j == m_InferBatchCountHorizontal - 1)
				{
					m_InferInputBatches[linearBatchIdx] = m_InferInput.slice_cols(batchOffset, inferLastBatchSizeHorizontal * m_InferBatchSizeVertical);
					m_InferOutputBatches[linearBatchIdx] = m_InferOutput.slice_cols(batchOffset, inferLastBatchSizeHorizontal * m_InferBatchSizeVertical);
					batchOffset += inferLastBatchSizeHorizontal * m_InferBatchSizeVertical;
				}
				else
				{
					m_InferInputBatches[linearBatchIdx] = m_InferInput.slice_cols(batchOffset, inferLastBatchSizeVertical * inferLastBatchSizeHorizontal);
					m_InferOutputBatches[linearBatchIdx] = m_InferOutput.slice_cols(batchOffset, inferLastBatchSizeVertical * inferLastBatchSizeHorizontal);
					batchOffset += inferLastBatchSizeVertical * inferLastBatchSizeHorizontal;
				}
			}
		}

		// Init train buffers
		if (m_TrainBatchSize % tcnn::BATCH_SIZE_GRANULARITY != 0) { en::Log::Error("NRC requires trainBatchSize to be a multiple of " + std::to_string(tcnn::BATCH_SIZE_GRANULARITY), true); }
		m_TrainInputBatches.resize(m_TrainMaxBatchLevel + 1);
		m_TrainTargetBatches.resize(m_TrainMaxBatchLevel + 1);

		for (uint32_t i = 0; i <= m_TrainMaxBatchLevel; i++)
		{
			const uint32_t levelSize = 1 << (i * 2);
			const uint32_t trainBatchCount = m_TrainBatchCountVertical * m_TrainBatchCountHorizontal / levelSize;
			const uint32_t batchSize = m_TrainBatchSize * levelSize;

			m_TrainInputBatches[i].resize(trainBatchCount);
			m_TrainTargetBatches[i].resize(trainBatchCount);

			for (uint32_t j = 0; j < trainBatchCount; j++)
			{
				m_TrainInputBatches[i][j] = m_TrainInput.slice_cols(j * batchSize, batchSize);
				m_TrainTargetBatches[i][j] = m_TrainTarget.slice_cols(j * batchSize, batchSize);
			}
		}

		en::Log::Info("Infer batch offset" + std::to_string(batchOffset) + ", infer count" + std::to_string(inferCount));
		en::Log::Info("Infer batch count (V:" + std::to_string(m_InferBatchCountVertical)+ ", H:" + std::to_string(m_InferBatchCountHorizontal) + ")");
		en::Log::Info("Infer batch size (V:" + std::to_string(m_InferBatchSizeVertical) + ", H:" + std::to_string(m_InferBatchSizeHorizontal) + ")");
		en::Log::Info("Train batch count (V:" + std::to_string(m_TrainBatchCountVertical) + ", H:" + std::to_string(m_TrainBatchCountHorizontal) + ")");
		en::Log::Info("Train batch size (V:" + std::to_string(m_TrainBatchSizeVertical) + ", H:" + std::to_string(m_TrainBatchSizeHorizontal) + ")");
	}

	void NeuralRadianceCache::InferAndTrain(const uint32_t* inferFilter, const uint32_t* trainFilter, bool train)
	{
		AwaitCudaStartSemaphore();

		auto start = std::chrono::steady_clock::now();
		Inference(inferFilter);
		auto end = std::chrono::steady_clock::now();
		double elapsed_ms = std::chrono::duration_cast<std::chrono::duration<double>>(end - start).count() * 1000.0;
		m_InferenceTime = elapsed_ms;

		if (train) { 
			auto start = std::chrono::steady_clock::now();
			Train(trainFilter); 
			auto end = std::chrono::steady_clock::now();
			double elapsed_ms = std::chrono::duration_cast<std::chrono::duration<double>>(end - start).count() * 1000.0;
			m_TrainTime = elapsed_ms;
		}

		SignalCudaFinishedSemaphore();
	}

	void NeuralRadianceCache::Destroy()
	{
	}

	float NeuralRadianceCache::GetLoss() const
	{
		return m_Loss;
	}

	float NeuralRadianceCache::GetInferenceTime() const
	{
		return m_InferenceTime;
	}

	float NeuralRadianceCache::GetTrainTime() const
	{
		return m_TrainTime;
	}

	size_t NeuralRadianceCache::GetInferBatchCount() const
	{
		return m_InferBatchCountVertical * m_InferBatchCountHorizontal;
	}

	size_t NeuralRadianceCache::GetTrainBatchCount() const
	{
		return m_TrainBatchCountVertical * m_TrainBatchCountHorizontal;
	}

	size_t NeuralRadianceCache::GetTrainBatchCountHorizontal() const
	{
		return m_TrainBatchCountHorizontal;
	}

	size_t NeuralRadianceCache::GetTrainBatchCountVertical() const
	{
		return m_TrainBatchCountVertical;
	}

	uint32_t NeuralRadianceCache::GetInferBatchSizeVertical() const
	{
		return m_InferBatchSizeVertical;
	}

	uint32_t NeuralRadianceCache::GetInferBatchSizeHorizontal() const
	{
		return m_InferBatchSizeHorizontal;
	}

	uint32_t NeuralRadianceCache::GetTrainBatchSizeVertical() const
	{
		return m_TrainBatchSizeVertical;
	}

	uint32_t NeuralRadianceCache::GetTrainBatchSizeHorizontal() const
	{
		return m_TrainBatchSizeHorizontal;
	}

	void NeuralRadianceCache::Inference(const uint32_t* inferFilter)
	{
		for (size_t i = 0; i < m_InferBatchCountVertical; i++)
		{
			for (int j = 0; j < m_InferBatchCountHorizontal; ++j)
			{
				const size_t linearBatchIndex = GetLinearInferBatchIndex(i, j);
				//en::Log::Info("Linear infer batch index " + std::to_string(linearBatchIndex)+ " has filter " + std::to_string(inferFilter[linearBatchIndex]));
				if (inferFilter[linearBatchIndex] > 0)
				{
					const tcnn::GPUMatrix<float>& inputBatch = m_InferInputBatches[linearBatchIndex];
					tcnn::GPUMatrix<float>& outputBatch = m_InferOutputBatches[linearBatchIndex];
					m_Model.network->inference(inputBatch, outputBatch);
				}
			}
		}
	}

	void NeuralRadianceCache::Train(const uint32_t* trainFilter)
	{
		std::vector<std::pair<uint32_t, uint32_t>> batchesToTrain;
		GetBatchesToTrain(m_TrainMaxBatchLevel, 0, 1 << (m_TrainMaxBatchLevel * 2), trainFilter, batchesToTrain);

		for (size_t i = 0; i < batchesToTrain.size(); i++)
		{
			const uint32_t batchLevel = batchesToTrain[i].first;
			const uint32_t batchIdx = batchesToTrain[i].second;

			const tcnn::GPUMatrix<float>& inputBatch = m_TrainInputBatches[batchLevel][batchIdx];
			const tcnn::GPUMatrix<float>& targetBatch = m_TrainTargetBatches[batchLevel][batchIdx];
			auto forwardContext = m_Model.trainer->training_step(inputBatch, targetBatch);
			m_Loss = m_Model.trainer->loss(*forwardContext.get());
		}
	}

	bool NeuralRadianceCache::GetBatchesToTrain(const int32_t currentBatchLevel, const uint32_t minBatchIdx, const uint32_t maxBatchIdx, const uint32_t* trainFilter, std::vector<std::pair<uint32_t, uint32_t>>& batchesToTrain)
	{
		if (currentBatchLevel < 0)
			return true;

		bool isFilterPositive = IsBatchFilterPositive(minBatchIdx, maxBatchIdx, trainFilter);
		if (!isFilterPositive)
		{
			return false;
		}

		const uint32_t size = (maxBatchIdx - minBatchIdx) / 4;
		
		bool p1 = GetBatchesToTrain(currentBatchLevel - 1, minBatchIdx, minBatchIdx + size, trainFilter, batchesToTrain);
		bool p2 = GetBatchesToTrain(currentBatchLevel - 1, minBatchIdx + size, minBatchIdx + size * 2, trainFilter, batchesToTrain);
		bool p3 = GetBatchesToTrain(currentBatchLevel - 1, minBatchIdx + size * 2, minBatchIdx + size * 3, trainFilter, batchesToTrain);
		bool p4 = GetBatchesToTrain(currentBatchLevel - 1, minBatchIdx + size * 3, minBatchIdx + size * 4, trainFilter, batchesToTrain);
		bool isAllChildPositive = p1 && p2 && p3 && p4;
		if (isAllChildPositive)
		{
			// If all subbatches are positive we can join them all in a single root batch
			if (currentBatchLevel == m_TrainMaxBatchLevel)
			{
				batchesToTrain.push_back({ m_TrainMaxBatchLevel, 0 });
			}
			
			return true;
		}

		const uint32_t childBatchLevel = currentBatchLevel - 1;
		const uint32_t childLevelSize = 1 << ((m_TrainMaxBatchLevel - childBatchLevel) * 2);
		if (p1)
		{
			const uint32_t currentBatchIdx = (minBatchIdx)/ childLevelSize;
			batchesToTrain.push_back({ childBatchLevel, currentBatchIdx });
		}

		if (p2)
		{
			const uint32_t currentBatchIdx = (minBatchIdx + size) / childLevelSize;
			batchesToTrain.push_back({ childBatchLevel, currentBatchIdx });
		}

		if (p3)
		{
			const uint32_t currentBatchIdx = (minBatchIdx + size * 2) / childLevelSize;
			batchesToTrain.push_back({ childBatchLevel, currentBatchIdx });
		}

		if (p4)
		{
			const uint32_t currentBatchIdx = (minBatchIdx + size * 3) / childLevelSize;
			batchesToTrain.push_back({ childBatchLevel, currentBatchIdx });
		}

		return false;
	}

	bool NeuralRadianceCache::IsBatchFilterPositive(const uint32_t minBatchIdx, const uint32_t maxBatchIdx, const uint32_t* trainFilter)
	{
		for (int i = minBatchIdx; i < maxBatchIdx; ++i)
		{
			if (trainFilter[i] > 0)
			{
				return true;
			}
		}

		return false;
	}

	void NeuralRadianceCache::AwaitCudaStartSemaphore()
	{
		hipExternalSemaphoreWaitParams extSemaphoreWaitParams;
		memset(&extSemaphoreWaitParams, 0, sizeof(extSemaphoreWaitParams));
		extSemaphoreWaitParams.params.fence.value = 0;
		extSemaphoreWaitParams.flags = 0;

		hipError_t error = hipWaitExternalSemaphoresAsync(&m_CudaStartSemaphore, &extSemaphoreWaitParams, 1);
		ASSERT_CUDA(error);
	}

	void NeuralRadianceCache::SignalCudaFinishedSemaphore()
	{
		hipExternalSemaphoreSignalParams extSemaphoreSignalParams;
		memset(&extSemaphoreSignalParams, 0, sizeof(extSemaphoreSignalParams));
		extSemaphoreSignalParams.params.fence.value = 0;
		extSemaphoreSignalParams.flags = 0;

		hipError_t error = hipSignalExternalSemaphoresAsync(&m_CudaFinishedSemaphore, &extSemaphoreSignalParams, 1);
		ASSERT_CUDA(error);
	}

	size_t NeuralRadianceCache::GetLinearInferBatchIndex(size_t verticalBatchIdx, size_t horizontalBatchIdx)
	{
		return verticalBatchIdx * m_InferBatchCountHorizontal + horizontalBatchIdx;
	}

	size_t NeuralRadianceCache::GetLinearTrainBatchIndex(size_t verticalBatchIdx, size_t horizontalBatchIdx)
	{
		return verticalBatchIdx * m_TrainBatchCountHorizontal + horizontalBatchIdx;
	}
}
