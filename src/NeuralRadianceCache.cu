#include "hip/hip_runtime.h"
#include <engine/cuda_common.hpp>
#include <engine/graphics/NeuralRadianceCache.hpp>
#include <random>
#include <engine/util/Log.hpp>

namespace en
{
	uint32_t NeuralRadianceCache::sc_InputCount = 5;
	uint32_t NeuralRadianceCache::sc_OutputCount = 3;

	NeuralRadianceCache::NeuralRadianceCache(const AppConfig& appConfig, const uint32_t renderWidth, const uint32_t renderHeight) :
		m_TrainBatchCountVertical(appConfig.trainBatchVerticalCount),
		m_TrainBatchCountHorizontal(appConfig.trainBatchHorizontalCount),
		m_InferBatchSize(2 << (appConfig.log2InferBatchSize - 1)),
		m_TrainBatchSize(2 << (appConfig.log2TrainBatchSize - 1)),
		m_InferBatchSizeVertical(sqrt(2 << (appConfig.log2InferBatchSize - 1))),
		m_InferBatchSizeHorizontal(sqrt(2 << (appConfig.log2InferBatchSize - 1))),
		m_InferBatchCountVertical(ceil((float) renderHeight / m_InferBatchSizeVertical)),
		m_InferBatchCountHorizontal(ceil((float) renderWidth / m_InferBatchSizeHorizontal)),
		m_TrainBatchSizeVertical(sqrt(2 << (appConfig.log2TrainBatchSize - 1))),
		m_TrainBatchSizeHorizontal(sqrt(2 << (appConfig.log2TrainBatchSize - 1)))
	{
		nlohmann::json modelConfig = {
			{"loss", {
				{"otype", appConfig.lossFn}
			}},
			{"optimizer", {
				{"otype", "EMA"},
				{"decay", appConfig.emaDecay},
				{"nested", {
					{"otype", appConfig.optimizer},
					{"learning_rate", appConfig.learningRate},
					//{"l2_reg", 0.0001},
				}}
			}},
			appConfig.encoding.jsonConfig,
			{"network", {
				{"otype", "FullyFusedMLP"},
				{"activation", "ReLU"},
				{"output_activation", "None"},
				{"n_neurons", appConfig.nnWidth},
				{"n_hidden_layers", appConfig.nnDepth},
			}},
		};

		m_Model = tcnn::create_from_config(sc_InputCount, sc_OutputCount, modelConfig);
	}

	void NeuralRadianceCache::Init(
		float renderWidth, 
		float renderHeight,
		float* dCuInferInput,
		float* dCuInferOutput,
		float* dCuTrainInput,
		float* dCuTrainTarget,
		hipExternalSemaphore_t cudaStartSemaphore,
		hipExternalSemaphore_t cudaFinishedSemaphore)
	{
		const uint32_t inferCount = renderWidth * renderHeight;

		// Check if sample counts are compatible
		if (inferCount % 16 != 0) { en::Log::Error("NRC requires inferCount to be a multiple of 16", true); }

		// Init members
		m_CudaStartSemaphore = cudaStartSemaphore;
		m_CudaFinishedSemaphore = cudaFinishedSemaphore;

		// Init big buffer
		const uint32_t trainCount = m_TrainBatchCountVertical * m_TrainBatchCountHorizontal * m_TrainBatchSizeVertical * m_TrainBatchSizeHorizontal;

		m_InferInput = tcnn::GPUMatrix<float>(dCuInferInput, sc_InputCount, inferCount);
		m_InferOutput = tcnn::GPUMatrix<float>(dCuInferOutput, sc_OutputCount, inferCount);
		m_TrainInput = tcnn::GPUMatrix<float>(dCuTrainInput, sc_InputCount, trainCount);
		m_TrainTarget = tcnn::GPUMatrix<float>(dCuTrainTarget, sc_OutputCount, trainCount);

		// Init infer buffers
		
		uint32_t inferBatchCount = m_InferBatchCountVertical * m_InferBatchCountHorizontal;
		if (m_InferBatchSize % tcnn::BATCH_SIZE_GRANULARITY != 0) { en::Log::Error("NRC requires inferBatchSize to be a multiple of " + std::to_string(tcnn::BATCH_SIZE_GRANULARITY), true); }
		m_InferInputBatches.resize(inferBatchCount);
		m_InferOutputBatches.resize(inferBatchCount);

		const uint32_t inferLastBatchSizeVertical = renderHeight - ((m_InferBatchCountVertical - 1) * m_InferBatchSizeVertical);
		const uint32_t inferLastBatchSizeHorizontal = renderWidth - ((m_InferBatchCountHorizontal - 1) * m_InferBatchSizeHorizontal);
		uint32_t batchOffset = 0;

		for (uint32_t i = 0; i < m_InferBatchCountVertical; i++)
		{
			for (uint32_t j = 0; j < m_InferBatchCountHorizontal; j++)
			{
				const uint32_t linearBatchIdx = GetLinearInferBatchIndex(i, j);
				if (i < m_InferBatchCountVertical - 1 && j < m_InferBatchCountHorizontal - 1)
				{
					m_InferInputBatches[linearBatchIdx] = m_InferInput.slice_cols(batchOffset, m_InferBatchSize);
					m_InferOutputBatches[linearBatchIdx] = m_InferOutput.slice_cols(batchOffset, m_InferBatchSize);
					batchOffset += m_InferBatchSize;
				}
				else if (i == m_InferBatchCountVertical - 1 && j < m_InferBatchCountHorizontal - 1)
				{
					m_InferInputBatches[linearBatchIdx] = m_InferInput.slice_cols(batchOffset, inferLastBatchSizeVertical * m_InferBatchSizeHorizontal);
					m_InferOutputBatches[linearBatchIdx] = m_InferOutput.slice_cols(batchOffset, inferLastBatchSizeVertical * m_InferBatchSizeHorizontal);
					batchOffset += inferLastBatchSizeVertical * m_InferBatchSizeHorizontal;
				}
				else if (i < m_InferBatchCountVertical - 1 && j == m_InferBatchCountHorizontal - 1)
				{
					m_InferInputBatches[linearBatchIdx] = m_InferInput.slice_cols(batchOffset, inferLastBatchSizeHorizontal * m_InferBatchSizeVertical);
					m_InferOutputBatches[linearBatchIdx] = m_InferOutput.slice_cols(batchOffset, inferLastBatchSizeHorizontal * m_InferBatchSizeVertical);
					batchOffset += inferLastBatchSizeHorizontal * m_InferBatchSizeVertical;
				}
				else
				{
					m_InferInputBatches[linearBatchIdx] = m_InferInput.slice_cols(batchOffset, inferLastBatchSizeVertical * inferLastBatchSizeHorizontal);
					m_InferOutputBatches[linearBatchIdx] = m_InferOutput.slice_cols(batchOffset, inferLastBatchSizeVertical * inferLastBatchSizeHorizontal);
					batchOffset += inferLastBatchSizeVertical * inferLastBatchSizeHorizontal;
				}
			}
		}

		// Init train buffers
		const uint32_t trainBatchCount = m_TrainBatchCountVertical * m_TrainBatchCountHorizontal;
		if (m_TrainBatchSize % tcnn::BATCH_SIZE_GRANULARITY != 0) { en::Log::Error("NRC requires trainBatchSize to be a multiple of " + std::to_string(tcnn::BATCH_SIZE_GRANULARITY), true); }
		m_TrainInputBatches.resize(trainBatchCount);
		m_TrainTargetBatches.resize(trainBatchCount);

		for (uint32_t i = 0; i < trainBatchCount; i++)
		{
			m_TrainInputBatches[i] = m_TrainInput.slice_cols(i * m_TrainBatchSize, m_TrainBatchSize);
			m_TrainTargetBatches[i] = m_TrainTarget.slice_cols(i * m_TrainBatchSize, m_TrainBatchSize);
		}

		en::Log::Info("Infer batch offset" + std::to_string(batchOffset) + ", infer count" + std::to_string(inferCount));
		en::Log::Info("Infer batch count (V:" + std::to_string(m_InferBatchCountVertical)+ ", H:" + std::to_string(m_InferBatchCountHorizontal) + ")");
		en::Log::Info("Infer batch size (V:" + std::to_string(m_InferBatchSizeVertical) + ", H:" + std::to_string(m_InferBatchSizeHorizontal) + ")");
		en::Log::Info("Train batch count (V:" + std::to_string(m_TrainBatchCountVertical) + ", H:" + std::to_string(m_TrainBatchCountHorizontal) + ")");
		en::Log::Info("Train batch size (V:" + std::to_string(m_TrainBatchSizeVertical) + ", H:" + std::to_string(m_TrainBatchSizeHorizontal) + ")");
	}

	void NeuralRadianceCache::InferAndTrain(const uint32_t* inferFilter, const uint32_t* trainFilter, uint32_t* trainFilteredFrameCounter, bool train)
	{
		AwaitCudaStartSemaphore();
		Inference(inferFilter);
		if (train) { Train(trainFilter, trainFilteredFrameCounter); }
		SignalCudaFinishedSemaphore();
	}

	void NeuralRadianceCache::Destroy()
	{
	}

	float NeuralRadianceCache::GetLoss() const
	{
		return m_Loss;
	}

	size_t NeuralRadianceCache::GetInferBatchCount() const
	{
		return m_InferBatchCountVertical * m_InferBatchCountHorizontal;
	}

	size_t NeuralRadianceCache::GetTrainBatchCount() const
	{
		return m_TrainBatchCountVertical * m_TrainBatchCountHorizontal;
	}

	size_t NeuralRadianceCache::GetTrainBatchCountHorizontal() const
	{
		return m_TrainBatchCountHorizontal;
	}

	size_t NeuralRadianceCache::GetTrainBatchCountVertical() const
	{
		return m_TrainBatchCountVertical;
	}

	uint32_t NeuralRadianceCache::GetInferBatchSizeVertical() const
	{
		return m_InferBatchSizeVertical;
	}

	uint32_t NeuralRadianceCache::GetInferBatchSizeHorizontal() const
	{
		return m_InferBatchSizeHorizontal;
	}

	uint32_t NeuralRadianceCache::GetTrainBatchSizeVertical() const
	{
		return m_TrainBatchSizeVertical;
	}

	uint32_t NeuralRadianceCache::GetTrainBatchSizeHorizontal() const
	{
		return m_TrainBatchSizeHorizontal;
	}

	void NeuralRadianceCache::Inference(const uint32_t* inferFilter)
	{
		for (size_t i = 0; i < m_InferBatchCountVertical; i++)
		{
			for (int j = 0; j < m_InferBatchCountHorizontal; ++j)
			{
				const size_t linearBatchIndex = GetLinearInferBatchIndex(i, j);
				//en::Log::Info("Linear infer batch index " + std::to_string(linearBatchIndex)+ " has filter " + std::to_string(inferFilter[linearBatchIndex]));
				if (inferFilter[linearBatchIndex] > 0)
				{
					const tcnn::GPUMatrix<float>& inputBatch = m_InferInputBatches[linearBatchIndex];
					tcnn::GPUMatrix<float>& outputBatch = m_InferOutputBatches[linearBatchIndex];
					m_Model.network->inference(inputBatch, outputBatch);
				}
			}
		}
	}

	void NeuralRadianceCache::Train(const uint32_t* trainFilter, uint32_t* trainFilteredFrameCounter)
	{
		for (size_t i = 0; i < m_TrainBatchCountVertical; i++)
		{
			for (size_t j = 0; j < m_TrainBatchCountHorizontal; j++)
			{
				const size_t linearBatchIndex = GetLinearTrainBatchIndex(i, j);
				//en::Log::Info("Linear train batch index " + std::to_string(linearBatchIndex) + " has filter " + std::to_string(trainFilter[linearBatchIndex]));

				if (trainFilter[linearBatchIndex] <= 0)
				{
					trainFilteredFrameCounter[linearBatchIndex] = std::min(trainFilteredFrameCounter[linearBatchIndex] + 1, sc_FilterFrameCountThreshold);
				}
				else
				{
					trainFilteredFrameCounter[linearBatchIndex] = 0;
				}

				// Exclude batch from training if it filtered more than sc_FilterFrameCountThreshold times
				// Batch is filtered if not a single ray scattered inside it
				if (trainFilteredFrameCounter[linearBatchIndex] < sc_FilterFrameCountThreshold)
				{
					const tcnn::GPUMatrix<float>& inputBatch = m_TrainInputBatches[linearBatchIndex];
					const tcnn::GPUMatrix<float>& targetBatch = m_TrainTargetBatches[linearBatchIndex];
					auto forwardContext = m_Model.trainer->training_step(inputBatch, targetBatch);
					m_Loss = m_Model.trainer->loss(*forwardContext.get());
				}
			}
		}
	}

	void NeuralRadianceCache::AwaitCudaStartSemaphore()
	{
		hipExternalSemaphoreWaitParams extSemaphoreWaitParams;
		memset(&extSemaphoreWaitParams, 0, sizeof(extSemaphoreWaitParams));
		extSemaphoreWaitParams.params.fence.value = 0;
		extSemaphoreWaitParams.flags = 0;

		hipError_t error = hipWaitExternalSemaphoresAsync(&m_CudaStartSemaphore, &extSemaphoreWaitParams, 1);
		ASSERT_CUDA(error);
	}

	void NeuralRadianceCache::SignalCudaFinishedSemaphore()
	{
		hipExternalSemaphoreSignalParams extSemaphoreSignalParams;
		memset(&extSemaphoreSignalParams, 0, sizeof(extSemaphoreSignalParams));
		extSemaphoreSignalParams.params.fence.value = 0;
		extSemaphoreSignalParams.flags = 0;

		hipError_t error = hipSignalExternalSemaphoresAsync(&m_CudaFinishedSemaphore, &extSemaphoreSignalParams, 1);
		ASSERT_CUDA(error);
	}

	size_t NeuralRadianceCache::GetLinearInferBatchIndex(size_t verticalBatchIdx, size_t horizontalBatchIdx)
	{
		return verticalBatchIdx * m_InferBatchCountHorizontal + horizontalBatchIdx;
	}

	size_t NeuralRadianceCache::GetLinearTrainBatchIndex(size_t verticalBatchIdx, size_t horizontalBatchIdx)
	{
		return verticalBatchIdx * m_TrainBatchCountHorizontal + horizontalBatchIdx;
	}
}
